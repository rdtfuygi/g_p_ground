#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES 
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "geometry.cuh"

#include "ground.cuh"

#include <opencv2/core/utils/logger.hpp>

#include <random>


#include <cmath>

#include "other.cuh"
#include "pipe.cuh"



void 建筑重置(std::vector<building>& b, double a_s)
{
	vector 初始解[8] = { vector(0.0,0.0),vector(0.0,-1.0),vector(0.0,-2.0),vector(1.0,-2.0),vector(1.0,-1.0),vector(-1.0,1.0),vector(-1.0,-1.0),vector(-1.0,0.0) };

	double 缩放 = (double(rand()) / RAND_MAX + 0.5);
	vector 平移(double(rand()) / RAND_MAX, double(rand()) / RAND_MAX);
	b = std::vector<building>(8);

	double s = 0;
	while (s < a_s)
	{
		s = 0;
		for (int i = 0; i < b.size(); i++)
		{
			b[i].target_area = double(rand()) / RAND_MAX * a_s * 0.5;
			s += b[i].target_area;
		}
	}

	for (int i = 0; i < b.size(); i++)
	{
		double 半径 = sqrt(b[i].target_area / M_PI) / 8;
		b[i].fun = i;
		for (int j = 0; j < 20; j++)
		{
			b[i].site[j].origin = point(vector(b[i].site[j].origin) + ((初始解[b[i].fun]) * 50) + ((vector(1.0, 0.0).rotate(18 * j + 45)) * 半径));
		}
		b[i].site.reset_seg();
	}
}



int main()
{
	srand(time(0));
	cv::utils::logging::setLogLevel(cv::utils::logging::LOG_LEVEL_ERROR);



	////m^3
	//std::vector<double>年需求量 = { 13950000,13950000 * 0.3,13950000 * 0.2 };
	//
	//std::vector<double>订货成本 = { 10000,10000,10000 };
	//std::vector<double>持有成本 = { 10,20,30 };
	//
	//std::vector<double>需求方差 = { 400,100,100 };
	//std::vector<double>提前期 = { 0.0004,0.0003,0.0003 };
	//std::vector<double>提前期方差 = { 0.003,0.003,0.003 };
	//std::vector<double>服务水平 = { 0.95,0.95,0.95 };
	//
	//std::vector<char>库存类型 = { 0,1,2 };
	//
	//std::vector<double>补货点_(年需求量.size());
	//std::vector<double>订货批量_(年需求量.size());
	//
	//
	//double 总需求 = 0;
	//for (int i = 0; i < 年需求量.size(); i++)
	//{
	//	补货点_[i] = 补货点(年需求量[i], 需求方差[i], 提前期[i], 提前期方差[i], 服务水平[i]);
	//	订货批量_[i] = 订货批量(年需求量[i], 订货成本[i], 持有成本[i]);
	//
	//
	//	总需求 += 年需求量[i];
	//}
	//
	//std::vector<double>仓库面积;
	//std::vector<double>仓库限高 = { 30,10,10 };
	//仓库面积_计算(仓库面积, 补货点_, 订货批量_, 库存类型, 仓库限高);
	//
	//std::vector<building> b;
	//
	//面积设定(b, 总需求, 仓库面积);


	int w = 900, h = 900;
	double 比例 = 1;
	point o(0, 0);


	pipe output_pip("asd_out", 8192);
	pipe input_pip("asd_in");
	pipe callback_pip("asd_back");


	wchar_t command[] = L"python D:\\Users\\57247\\OneDrive\\Desktop\\g_p\\g_p_ground_ai\\g_p_ground_ai.py asd";
	STARTUPINFOW si = { 0 };
	PROCESS_INFORMATION pi = { 0 };
	CreateProcessW
	(
		NULL,
		command,
		NULL,
		NULL,
		FALSE,
		NULL,
		NULL,
		NULL,
		&si,
		&pi
	);

	//vector 初始解[8] = { vector(0.0,0.0),vector(0.0,-1.0),vector(0.0,-2.0),vector(1.0,-2.0),vector(1.0,-1.0),vector(-1.0,1.0),vector(-1.0,-1.0),vector(-1.0,0.0) };

	int loops = 0;



	while(true)
	{
		ground a;
		cv::Mat p;
		while (true)
		{
			a = 场地设定(double(rand()) / RAND_MAX * 200000 + 200000);
			a.site.move2center();

			p = cv::Mat::zeros(h, w, CV_8UC3);
			a.print(p, 比例, cv::Scalar(255, 255, 255));

			cv::imshow("123", p);
			cv::waitKey(1000);
			if (cv::waitKey(3000) != 'n')
			{
				break;
			}
		}

		double a_s = a.area();

		std::vector<building> b(8);



		建筑重置(b, a_s);

		for (int i = 0; i < b.size(); i++)
		{
			b[i].print(p, 比例, cv::Scalar(255, 255 / b.size() * i, 0));
		}

		cv::imshow("123", p);
		cv::waitKey(1);

		int r = 0;

		double 分数;
		{
			bool temp;
			分数 = 奖励函数(a, b, temp);
		}

		while(true)
		{
			std::vector<double> output;
			output.reserve(936);
			std::vector<double> a_data;
			a.data(a_data);
			output.insert(output.end(), a_data.begin(), a_data.end());
			for (int i = 0; i < b.size(); i++)
			{
				std::vector<double> b_data;
				b[i].data(b_data);
				output.insert(output.end(), b_data.begin(), b_data.end());
			}

			output_pip.send(output);

			std::vector<double> input;
			input_pip.receive(input);

			
			for (int i = 0; i < b.size(); i++)
			{
				std::vector<vector> input_point;
				input_point.reserve(20);
				for (int j = 0; j < 20; j++)
				{				
					vector temp = vector(point(input[i * 42 + j * 2], input[i * 42 + j * 2 + 1]));
					input_point.push_back(temp);
				}
				b[i].move(input_point.data());
				b[i].door[0] = int(input[i * 42 + 40]);
				b[i].door[1] = int(input[i * 42 + 41]);
				b[i].site.reset_seg();
				//b[i].site.simple(5);
			}
			std::vector<double> callback;
			bool reset = false;
			callback.reserve(2);

			{
				double temp = 奖励函数(a, b, reset);
				callback.push_back(temp - 分数);
				分数 = temp;
			}
			callback_pip.send(callback);

			if (reset)
			{
				r += 10;
			}
			if (r > 1000)
			{
				r = 0;
				建筑重置(b, a_s);
				{
					bool temp;
					分数 = 奖励函数(a, b, temp);
				}
			}
			r = (r > 0) ? r - 5 : 0;

			if ((loops % 1) == 0)
			{
				p = cv::Mat::zeros(h, w, CV_8UC3);
				a.print(p, 比例, cv::Scalar(255, 255, 255));
				for (int i = 0; i < b.size(); i++)
				{
					b[i].print(p, 比例, cv::Scalar(255, 255 / b.size() * i, 0));
				}


				cv::imshow("123", p);
				int key = cv::waitKey(1);
				if (key == 'n')
				{
					loops = 0;
					break;
				}
				else if (key == 'r')
				{
					建筑重置(b, a_s);
					{
						bool temp;
						分数 = 奖励函数(a, b, temp);
					}
				}
			}

			double s = 0;
			for (int i = 0; i < b.size(); i++)
			{
				s += b[i].area();
			}
			if (s > a_s)
			{
				建筑重置(b, a_s);
				{
					bool temp;
					分数 = 奖励函数(a, b, temp);
				}
			}

			loops++;
			if (loops >= 10000)
			{
				loops = 0;
				break;
			}
		}
	}

	CloseHandle(pi.hProcess);
	CloseHandle(pi.hThread);

	return 0;
}