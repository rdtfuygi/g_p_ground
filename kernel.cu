#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "geometry.cuh"

#include "ground.cuh"

#include <opencv2/core/utils/logger.hpp>

#include <random>


#include <cmath>

#include "other.cuh"

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main_()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}





int main()
{
	srand(time(0));
	cv::utils::logging::setLogLevel(cv::utils::logging::LOG_LEVEL_ERROR);



	//m^3
	std::vector<double>年需求量 = { 13950000 };

	std::vector<double>订货成本 = { 10000 };
	std::vector<double>持有成本 = { 10 };

	std::vector<double>需求方差 = { 400 };
	std::vector<double>提前期 = { 0.0004 };
	std::vector<double>提前期方差 = { 0.003 };
	std::vector<double>服务水平 = { 0.95 };

	std::vector<char>库存类型 = { 0 };

	std::vector<double>补货点_(年需求量.size());
	std::vector<double>订货批量_(年需求量.size());


	double 总需求 = 0;
	for (int i = 0; i < 年需求量.size(); i++)
	{
		补货点_[i] = 补货点(年需求量[i], 需求方差[i], 提前期[i], 提前期方差[i], 服务水平[i]);
		订货批量_[i] = 订货批量(年需求量[i], 订货成本[i], 持有成本[i]);


		总需求 += 年需求量[i];
	}

	std::vector<double>仓库面积;
	std::vector<double>仓库限高 = { 30,10,10 };
	仓库面积_计算(仓库面积, 补货点_, 订货批量_, 库存类型, 仓库限高);

	std::vector<building> b;

	面积设定(b, 总需求, 仓库面积);


	int w = 1500, h = 900;
	double 比例 = 1;
	point o(0, 0);


	ground abc;
	while(true)
	{
		for (int i = 0; i < 3; i++)
		{
			abc.site[i] = seg((double(rand()) / RAND_MAX) * 800, (double(rand()) / RAND_MAX) * 800, (double(rand()) / RAND_MAX) * 800, (double(rand()) / RAND_MAX) * 800);
		}
		abc.site.reset_seg();

		cv::Mat a = cv::Mat::zeros(h, w, CV_8UC3);
		abc.print(a, 比例, cv::Scalar(255, 255, 255));




		abc.print(a, 比例, cv::Scalar(255, 255, 255));


		cv::imshow("123", a);
		cv::waitKey(1);

		double s = abc.area();

		printf("%3f\n", s);

		if (abc.site.legal() && (s > 400000))
		{
			break;
		}
	}




	abc.site.move2center();

	cv::Mat a = cv::Mat::zeros(h, w, CV_8UC3);
	abc.print(a, 比例, cv::Scalar(255, 255, 255));

	cv::imshow("123", a);
	cv::waitKey(0);


	double 温度 = 10000;
	double 冷却_率 = 0.999;
	double g = -DBL_MAX;
	while (温度 > 1)
	{
		std::vector<building> b_t = b;
		for (int i = 1; i < b_t.size(); i++)
		{
			for (int j = 0; j < 20; j++)
			{
				b_t[i].site[j].origin = point(vector(b_t[i].site[j].origin) + vector((double(rand()) / RAND_MAX - 0.5) * 2, (double(rand()) / RAND_MAX - 0.5) * 2));
			}
			b_t[i].site.reset_seg();
		}
		b_t[0] = 停车场设置(b_t[1]);
		double g_t = 奖励函数(abc, b_t);
		if (g < g_t)
		{
			b = b_t;
			g = g_t;
		}
		else
		{
			double d = g_t - g;
			if (exp(-d / 温度) > (double(rand()) / RAND_MAX))
			{
				b = b_t;
				g = g_t;
			}
		}
		温度 *= 冷却_率;

		cv::Mat a = cv::Mat::zeros(h, w, CV_8UC3);
		abc.print(a, 比例, cv::Scalar(255, 255, 255));
		for (int i = 0; i < b.size(); i++)
		{
			b[i].print(a, 比例, cv::Scalar(0, 255 / b.size() * i, 0));
		}

		cv::imshow("123", a);
		cv::waitKey(1);
	}
	

	return 0;
}