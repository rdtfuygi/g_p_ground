#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "geometry.cuh"

#include "ground.cuh"

#include <opencv2/core/utils/logger.hpp>

#include <random>


#include <cmath>

#include "other.cuh"
#include "pipe.cuh"

//hipError_t addWithCuda(int *c, const int *p, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *p, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = p[i] + b[i];
//}
//
//int main_()
//{
//    const int arraySize = 5;
//    const int p[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, p, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *p, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on p multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have p CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, p, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch p kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}





int main()
{
	srand(time(0));
	cv::utils::logging::setLogLevel(cv::utils::logging::LOG_LEVEL_ERROR);



	////m^3
	//std::vector<double>年需求量 = { 13950000,13950000 * 0.3,13950000 * 0.2 };
	//
	//std::vector<double>订货成本 = { 10000,10000,10000 };
	//std::vector<double>持有成本 = { 10,20,30 };
	//
	//std::vector<double>需求方差 = { 400,100,100 };
	//std::vector<double>提前期 = { 0.0004,0.0003,0.0003 };
	//std::vector<double>提前期方差 = { 0.003,0.003,0.003 };
	//std::vector<double>服务水平 = { 0.95,0.95,0.95 };
	//
	//std::vector<char>库存类型 = { 0,1,2 };
	//
	//std::vector<double>补货点_(年需求量.size());
	//std::vector<double>订货批量_(年需求量.size());
	//
	//
	//double 总需求 = 0;
	//for (int i = 0; i < 年需求量.size(); i++)
	//{
	//	补货点_[i] = 补货点(年需求量[i], 需求方差[i], 提前期[i], 提前期方差[i], 服务水平[i]);
	//	订货批量_[i] = 订货批量(年需求量[i], 订货成本[i], 持有成本[i]);
	//
	//
	//	总需求 += 年需求量[i];
	//}
	//
	//std::vector<double>仓库面积;
	//std::vector<double>仓库限高 = { 30,10,10 };
	//仓库面积_计算(仓库面积, 补货点_, 订货批量_, 库存类型, 仓库限高);
	//
	//std::vector<building> b;
	//
	//面积设定(b, 总需求, 仓库面积);


	int w = 900, h = 900;
	double 比例 = 1;
	point o(0, 0);


	pipe output_pip("asd_out", 8192);
	pipe input_pip("asd_in");
	pipe callback_pip("asd_back");


	wchar_t command[] = L"python D:\\Users\\57247\\OneDrive\\Desktop\\g_p\\g_p_ground_ai\\g_p_ground_ai.py asd";
	STARTUPINFOW si = { 0 };
	PROCESS_INFORMATION pi = { 0 };
	CreateProcessW
	(
		NULL,
		command,
		NULL,
		NULL,
		FALSE,
		NULL,
		NULL,
		NULL,
		&si,
		&pi
	);

	vector 初始解[8] = { vector(0.0,0.0),vector(0.0,-1.0),vector(0.0,-2.0),vector(1.0,-2.0),vector(1.0,-1.0),vector(-1.0,1.0),vector(-1.0,-1.0),vector(-1.0,0.0) };

	while(true)
	{
		ground a;
		cv::Mat p;
		while (true)
		{
			a = 场地设定(double(rand()) / RAND_MAX * 200000 + 200000);
			a.site.move2center();

			p = cv::Mat::zeros(h, w, CV_8UC3);
			a.print(p, 比例, cv::Scalar(255, 255, 255));

			cv::imshow("123", p);
			cv::waitKey(1000);
			if (cv::waitKey(3000) != 'r')
			{
				break;
			}
		}

		double s = a.area();

		std::vector<building> b(8);



		for (int i = 0; i < b.size(); i++)
		{
			b[i].fun = i;
			for (int j = 0; j < 20; j++)
			{
				b[i].site[j].origin = point(vector(b[i].site[j].origin) + ((初始解[b[i].fun]) * 50) + ((vector(1.0, 0.0).rotate(18 * j + 45)) * 10));
			}


			b[i].site.reset_seg();

			b[i].target_area = double(rand()) / RAND_MAX * s * 0.125;
		}

		//double 温度 = 10000;
		//double 冷却_率 = 0.999;
		//double g = -DBL_MAX;
		//while (温度 > 1)
		//{
		//	std::vector<building> b_t = b;
		//	for (int i = 0; i < b_t.size(); i++)
		//	{
		//		for (int j = 0; j < 20; j++)
		//		{
		//			b_t[i].site[j].origin = point(vector(b_t[i].site[j].origin) + vector((double(rand()) / RAND_MAX - 0.5) * 2, (double(rand()) / RAND_MAX - 0.5) * 2));
		//		}
		//
		//		b_t[i].site.reset_seg();
		//		//b_t[0] = 停车场设置(b_t[1]);
		//		double g_t = 奖励函数(a, b_t);
		//		printf("%3f\n", g_t);
		//		if (g < g_t)
		//		{
		//			b = b_t;
		//			g = g_t;
		//		}
		//		else
		//		{
		//			double d = g_t - g;
		//			if (exp(-d / 温度) > (double(rand()) / RAND_MAX))
		//			{
		//				b = b_t;
		//				g = g_t;
		//			}
		//		}
		//		cv::Mat p = cv::Mat::zeros(h, w, CV_8UC3);
		//		a.print(p, 比例, cv::Scalar(255, 255, 255));
		//		for (int i = 0; i < b.size(); i++)
		//		{
		//			b[i].print(p, 比例, cv::Scalar(255, 255 / b.size() * i, 0));
		//		}
		//		cv::imshow("123", p);
		//		cv::waitKey(1);
		//	}
		//	温度 *= 冷却_率;
		//}

		for (int i = 0; i < b.size(); i++)
		{
			b[i].print(p, 比例, cv::Scalar(255, 255 / b.size() * i, 0));
		}

		cv::imshow("123", p);
		cv::waitKey(1);

		while(true)
		{
			std::vector<double> output;
			output.reserve(926);
			std::vector<double> a_data;
			a.data(a_data);
			output.insert(output.end(), a_data.begin(), a_data.end());
			for (int i = 0; i < b.size(); i++)
			{
				std::vector<double> b_data;
				b[i].data(b_data);
				output.insert(output.end(), b_data.begin(), b_data.end());
			}

			output_pip.send(output);

			std::vector<double> input;
			input_pip.receive(input);

			
			for (int i = 0; i < b.size(); i++)
			{
				std::vector<vector> input_point;
				input_point.reserve(20);
				for (int j = 0; j < 20; j++)
				{				
					vector temp = vector(point(input[i * 42 + j * 2], input[i * 42 + j * 2 + 1])) * 5;
					input_point.push_back(temp);
				}
				b[i].move(input_point.data());
				b[i].door[0] = int(input[i * 42 + 40]);
				b[i].door[1] = int(input[i * 42 + 41]);
				b[i].site.reset_seg();
			}
			std::vector<double> callback(1);
			callback[0] = 奖励函数(a, b);
			callback_pip.send(callback);

			p = cv::Mat::zeros(h, w, CV_8UC3);
			a.print(p, 比例, cv::Scalar(255, 255, 255));
			for (int i = 0; i < b.size(); i++)
			{
				b[i].print(p, 比例, cv::Scalar(255, 255 / b.size() * i, 0));
			}


			cv::imshow("123", p);
			int key = cv::waitKey(1);
			if (key == 'n')
			{
				break;
			}
			else if (key == 'r')
			{
				b = std::vector<building>(8);
				for (int i = 0; i < b.size(); i++)
				{
					b[i].fun = i;
					for (int j = 0; j < 20; j++)
					{
						b[i].site[j].origin = point(vector(b[i].site[j].origin) + ((初始解[b[i].fun]) * 50) + ((vector(1.0, 0.0).rotate(18 * j + 45)) * 10));
					}


					b[i].site.reset_seg();

					b[i].target_area = double(rand()) / RAND_MAX * s * 0.125;
				}
			}

			double s = 0;
			for (int i = 0; i < b.size(); i++)
			{
				s += b[i].area();
			}
			if (s > a.area())
			{
				b = std::vector<building>(8);
				for (int i = 0; i < b.size(); i++)
				{
					b[i].fun = i;
					for (int j = 0; j < 20; j++)
					{
						b[i].site[j].origin = point(vector(b[i].site[j].origin) + ((初始解[b[i].fun]) * 50) + ((vector(1.0, 0.0).rotate(18 * j + 45)) * 10));
					}


					b[i].site.reset_seg();

					b[i].target_area = double(rand()) / RAND_MAX * s * 0.125;
				}
			}
		}
	}

	return 0;
}