#include "hip/hip_runtime.h"
#include"ground.cuh"

//#define _USE_MATH_DEFINES 
//#include <cmath>

__host__ __device__ ground::ground() :site(), door() {}

__host__ __device__ ground::ground(point ��[20], int ��_1, int ��_2) : site(��)
{
	door[0] = ��_1;
	door[1] = ��_2;
}

ground::ground(std::vector<point>& ��, int ��_1, int ��_2) : site(��)
{
	door[0] = ��_1;
	door[1] = ��_2;
}

__host__ __device__ seg ground::get_door(int i) const
{
	return site[door[i]];
}

__host__ __device__ double ground::area() const
{
	return site.area();
}

void ground::print(cv::InputOutputArray ͼ��, double ����, const cv::Scalar& ��ɫ, int ��ϸ) const
{
	site.print(ͼ��, ����, ��ɫ, ��ϸ);
	get_door(0).print(ͼ��, ����, ��ɫ, ��ϸ * 2);
	get_door(1).print(ͼ��, ����, ��ɫ, ��ϸ * 2);
	//site.center().print(ͼ��, ����, ��ɫ, ��ϸ);
}

void ground::data(std::vector<double>& ����)
{
	���� = std::vector<double>();
	����.reserve(103);
	for (int i = 0; i < 20; i++)
	{
		����.push_back(site[i].origin[0]);
		����.push_back(site[i].origin[1]);
		����.push_back(site[i].dir[0]);
		����.push_back(site[i].dir[1]);
		����.push_back(site[i].dist);
	}
	����.push_back(double(door[0]));
	����.push_back(double(door[1]));
	����.push_back(area());
}


building::building() :ground(), fun(0), target_area(0) {}

building::building(point ��[20], int ��_1, int ��_2, int ����, double Ŀ���С) :ground(��, ��_1, ��_2), fun(����), target_area(Ŀ���С) {}

building::building(std::vector<point>& ��, int ��_1, int ��_2, int ����, double Ŀ���С) :ground(��, ��_1, ��_2), fun(����), target_area(Ŀ���С) {}

__host__ __device__ void building::move(vector �ƶ�, int index)
{
	site[index].origin = point(vector(site[index].origin) + �ƶ�);
}

__host__ __device__ void building::move(vector �ƶ�[20])
{
	for (int i = 0; i < 20; i++)
	{
		move(�ƶ�[i], i);
	}
	site.reset_seg();
}

void building::move(std::vector<vector>& �ƶ�, std::vector<building>& b, ground a, int& n)
{
	for (int i = 0; i < 20; i++)
	{
		poly temp = site;
		move(�ƶ�[i], i);

		site.reset_seg();

		bool m = true;

		if (((site[(i + 1) % 20].dir * site[i % 20].dir) < -M_SQRT1_2) || ((site[i].dir * site[(i + 19) % 20].dir) < -M_SQRT1_2) || ((site[(i + 19) % 20].dir * site[(i + 18) % 20].dir) < -M_SQRT1_2))
		{
			m = false;
		}
		else if (!site.legal())
		{
			m = false;
		}
		else if (!a.site.full_overlap(site))
		{
			m = false;
		}
		else
		{
			for (int j = 0; j < 8; j++)
			{
				if(j!=fun)
				{
					if (site.is_overlap(b[j].site))
					{
						m = false;
						break;
					}
				}
			}
		}



		if (!m)
		{
			n++;
			site = temp;
		}
	}
}

__host__ __device__ void building::change(point ��, int index)
{
	site[index].origin = ��;
}

__host__ __device__ void building::change(point ��[20])
{
	for (int i = 0; i < 20; i++)
	{
		change(��[i], i);
	}
	site.reset_seg();
}

void building::data(std::vector<double>& ����)
{
	���� = std::vector<double>();
	����.reserve(106);
	point ���� = site.center();
	����.push_back(����[0] / 1024);
	����.push_back(����[1] / 1024);
	for (int i = 0; i < 20; i++)
	{
		����.push_back(site[i].origin[0]);
		����.push_back(site[i].origin[1]);
		����.push_back(site[i].dir[0]);
		����.push_back(site[i].dir[1]);
		����.push_back(site[i].dist);
	}
	����.push_back(double(door[0]));
	����.push_back(double(door[1]));
	����.push_back(area());
	����.push_back(target_area);
}




//__global__ void building_move(building* ����, vector* �ƶ�, int �ߴ�)
//{
//	int i = threadIdx.x + blockIdx.x * blockDim.x;
//	if (i >= �ߴ�)
//	{
//		return;
//	}
//	����[i / 20].move(�ƶ�[i], i % 20);
//}
//
//__global__ void building_reset_seg(building* ����, int �ߴ�)
//{
//	int i = threadIdx.x + blockIdx.x * blockDim.x;
//	if (i >= �ߴ�)
//	{
//		return;
//	}
//	����[i / 20].site.reset_seg(i % 20);
//}
//
//void ��������(std::vector<building>& ����, std::vector<vector>& �ƶ�)
//{
//	int cuda�豸����;
//	hipGetDeviceCount(&cuda�豸����);
//	if (cuda�豸���� == 0)
//	{
//		for (int i = 0; i < ����.size(); i++)
//		{
//			for (int j = 0; j < �ƶ�.size(); j++)
//			{
//				����[i].move(�ƶ�[i * 20 + j], j);
//			}
//			����[i].site.reset_seg();
//		}
//	}
//	else
//	{
//		int �Կ�id;
//		hipGetDevice(&�Կ�id);
//		hipDeviceProp_t �Կ�����;
//		hipGetDeviceProperties(&�Կ�����, �Կ�id);
//		int ÿ���߳� = �Կ�����.maxThreadsPerBlock;
//		int ���� = �ƶ�.size() / ÿ���߳� + 1;
//
//		building* ����_dev = NULL;//
//		vector* �ƶ�_dev = NULL;//
//		hipMalloc((void**)����_dev, sizeof(building) * ����.size());
//		hipMalloc((void**)�ƶ�_dev, sizeof(vector) * �ƶ�.size());
//		hipMemcpy(����_dev, ����.data(), sizeof(building) * ����.size(), hipMemcpyHostToDevice);
//		hipMemcpy(�ƶ�_dev, ����.data(), sizeof(vector) * �ƶ�.size(), hipMemcpyHostToDevice);
//
//		building_move << < ����, ÿ���߳� >> > (����_dev, �ƶ�_dev, �ƶ�.size());
//
//		hipFree(�ƶ�_dev);
//
//		building_reset_seg << < ����, ÿ���߳� >> > (����_dev, �ƶ�.size());
//
//		hipMemcpy(����.data(), ����_dev, sizeof(building) * ����.size(), hipMemcpyDeviceToHost);
//		hipFree(����_dev);
//	}
//}

building ͣ��������(building �ּ���)
{
	seg ƽ��[5];
	double ��ת = 90;
	if (�ּ���.site.dir_area() > 0)
	{
		��ת = -90;
	}

	for (int i = 0; i < 5; i++)
	{
		ƽ��[i].origin = rotate(�ּ���.site[i].origin, �ּ���.site[i].point_get(27), ��ת);
		ƽ��[i].dir = �ּ���.site[i].dir;
		ƽ��[i].dist = �ּ���.site[i].dist;
	}



	point ��[20];
	for (int i = 0; i < 5; i++)
	{
		��[i] = �ּ���.site[i].origin;
	}
	��[5] = ƽ��[4].end();
	for (int i = 1; i < 4; i++)
	{
		��[i + 5] = cross(line(ƽ��[5 - i]), line(ƽ��[5 - i - 1]));
	}
	��[9] = ƽ��[0].origin;

	for (int i = 10; i < 20; i++)
	{
		��[i] = ��[0];
	}


	return building(��, 0, 4, fun_port, 0);
}



const char ������[8][8] =
{
	{0, 4, 0, 0, 0, 3, 0, 0},
	{4, 0, 4, 2, 2, 0, 0, 0},
	{0, 4, 0, 0,-1, 0, 0, 0},
	{0, 2, 0, 0,-1, 0, 0, 0},
	{0, 2,-1,-1, 0,-1,-1, 0},
	{3, 0, 0, 0,-1, 0, 3, 1},
	{0, 0, 0, 0,-1, 3, 0, 4},
	{0, 0, 0, 0, 0, 1, 4, 0}
};

double
���_Ȩ�� = 0,
ƽֱ��_Ȩ�� = 0,
����_Ȩ�� = 0,
��_Ȩ�� = 0,
�ܳ�_Ȩ�� = 0;

double ��������(ground ����, std::vector<building>& ����, bool& reset)
{
	double ���� = 0;




	for (int i = 0; i < ����.size(); i++)
	{
		double ��� = ����[i].area();

		//if (����.site.full_overlap(����[i].site))
		//{
		//	���� += ������_Ȩ��;
		//}
		//else
		//{
		//	double a = fmin(1, pow(overlap_area(����.site, ����[i].site) / ���, 2));
		//	���� += ������_Ȩ�� * a / 2;
		//	reset = true;
		//}

		for (int j = 0; j < i; j++)
		{
			//if (!����[i].site.is_overlap(����[j].site))
			//{
			//	���� += �ص�_Ȩ�� / 28 * 8;
			//}
			//else
			//{
			//	double a = (1 - fmin(1, pow(overlap_area(����[j].site, ����[i].site) / ���, 2)));
			//	���� += �ص�_Ȩ�� * a / 2 / 28 * 8;
			//	reset = true;
			//}

			if (������[����[i].fun][����[j].fun] >= 0)
			{
				���� += exp(-dist(����[i].site, ����[j].site) * ������[����[i].fun][����[j].fun] / 100) * ����_Ȩ�� / 28 * 8;

				���� += exp(-fmin(dist(����[j].site, ����[i].get_door(0)), dist(����[j].site, ����[i].get_door(1))) * ������[����[i].fun][����[j].fun] / 100) * ��_Ȩ�� / 28 * 8;
			}
			else
			{
				���� += (1 - exp(-dist(����[i].site, ����[j].site)) / 100) * ����_Ȩ�� / 28 * 8;

				���� += (1 - exp(-fmin(dist(����[j].site, ����[i].get_door(0)), dist(����[j].site, ����[i].get_door(1))) / 100)) * ��_Ȩ�� / 28 * 8;
			}
		}

		���� += -pow((����[i].target_area - ���) / 1048576, 2) * ���_Ȩ��;

		double �ܳ� = 0;
		for (int j = 0; j < 20; j++)
		{
			double a = fmax(fmax((����[i].site[j].dir * ����[i].site[(j + 1) % 20].dir), 0), abs(����[i].site[j].dir ^ ����[i].site[(j + 1) % 20].dir));
			���� += (a + pow(a, 16)) / 2 / 20 * ƽֱ��_Ȩ��;

			�ܳ� += ����[i].site[j].dist;

			if (a < M_SQRT1_2)
			{
				reset = true;
			}
		}

		���� += (4 * sqrt(���) - fmax(�ܳ�, 4 * sqrt(���))) / 1024 * �ܳ�_Ȩ��;

		//if (����[i].site.legal())
		//{
		//	���� += �Ϸ�_Ȩ��;
		//}
		//else
		//{
		//	double a = fmin(1, pow(����[i].site.dir_area() / ���, 2));
		//	���� += �Ϸ�_Ȩ�� * a / 2;
		//	reset = true;
		//}
	}
	return ���� / 8;
}

void �ֿ����_����(std::vector<double>& �ֿ����, std::vector<double>& ������_, std::vector<double>& ��������_, std::vector<char>& �������, std::vector<double>& �ֿ��޸�)
{
	�ֿ���� = { 0,0,0 };
	for (int i = 0; i < ������_.size(); i++)
	{
		�ֿ����[�������[i]] += (������_[i] + ��������_[i]) / �ֿ��޸�[�������[i]];
	}
}

void ����趨(std::vector<building>& ����, double ������, std::vector<double>& �ֿ����)
{
	���� = std::vector<building>(8);

	����[0].target_area = ������ / 30 / 365 / 24 * 3 * 35;
	����[1].target_area = ������ / 30 / 365 / 24 * 3 * 20;
	����[2].target_area = �ֿ����[0] * 1.7;
	����[3].target_area = �ֿ����[1] * 1.7;
	����[4].target_area = �ֿ����[2] * 1.7;
	����[5].target_area = 5000;
	����[6].target_area = ������ / 30 / 365 / 24 * (3 + 2) * 10;
	����[7].target_area = (500 * 0.7 + ������ / 30 / 365 / 24 * 3 * 0.3) * 40;

	for (int i = 0; i < ����.size(); i++)
	{
		����[i].fun = i;
	}
}
