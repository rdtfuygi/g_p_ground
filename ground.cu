#include "hip/hip_runtime.h"
#include"ground.cuh"

//#define _USE_MATH_DEFINES 
//#include <cmath>

__host__ __device__ ground::ground() :site(), door() {}

__host__ __device__ ground::ground(point ��[20], int ��_1, int ��_2) : site(��)
{
	door[0] = ��_1;
	door[1] = ��_2;
}

ground::ground(std::vector<point>& ��, int ��_1, int ��_2) : site(��)
{
	door[0] = ��_1;
	door[1] = ��_2;
}

__host__ __device__ seg ground::get_door(int i) const
{
	return site[door[i]];
}

__host__ __device__ double ground::area() const
{
	return site.area();
}

void ground::print(cv::InputOutputArray ͼ��, double ����, const cv::Scalar& ��ɫ, int ��ϸ) const
{
	site.print(ͼ��, ����, ��ɫ, ��ϸ);
	get_door(0).print(ͼ��, ����, ��ɫ, ��ϸ * 2);
	get_door(1).print(ͼ��, ����, ��ɫ, ��ϸ * 2);
	//site.center().print(ͼ��, ����, ��ɫ, ��ϸ);
}

void ground::data(std::vector<double>& ����)
{
	���� = std::vector<double>();
	����.reserve(103);
	for (int i = 0; i < 20; i++)
	{
		����.push_back(site[i].origin[0]);
		����.push_back(site[i].origin[1]);
		����.push_back(site[i].dir[0]);
		����.push_back(site[i].dir[1]);
		����.push_back(site[i].dist);
	}
	����.push_back(area());
	����.push_back(double(door[0]));
	����.push_back(double(door[1]));

}


building::building() :ground(), fun(0), target_area(0) {}

building::building(point ��[20], int ��_1, int ��_2, int ����, double Ŀ���С) :ground(��, ��_1, ��_2), fun(����), target_area(Ŀ���С) {}

building::building(std::vector<point>& ��, int ��_1, int ��_2, int ����, double Ŀ���С) :ground(��, ��_1, ��_2), fun(����), target_area(Ŀ���С) {}

__host__ __device__ void building::move(vector �ƶ�, int index)
{
	site[index].origin = point(vector(site[index].origin) + �ƶ�);
}

__host__ __device__ void building::move(vector �ƶ�[20])
{
	for (int i = 0; i < 20; i++)
	{
		move(�ƶ�[i], i);
	}
	site.reset_seg();
}

void building::move(vector �ƶ�[20], std::vector<building>& b, ground a)
{
	for (int i = 0; i < 20; i++)
	{
		seg t[3] =
		{
			seg(point(vector(site[i].origin) + �ƶ�[i]),site[(i + 1) % 20].origin),
			seg(point(vector(site[i].origin) + �ƶ�[i]),site[(i + 19) % 20].origin),
			seg(point(vector(site[i].origin) + �ƶ�[i]),site[i].origin)
		};
		
		bool m = true;
		for (int j = 0; j < 8; j++)
		{
			for (int k = 0; k < 20; k++)
			{
				if ((fun == j) && ((k == i) || (k == ((i + 19) % 20))))
				{
					if (((t[0].dir * t[1].dir) > 0.99) || ((t[0].dir * b[j].site[(i + 1) % 20].dir) < -0.99) || ((b[j].site[(i + 18) % 20].dir * t[1].dir) > 0.99))
					{
						m = false;
						break;
					}
					else
					{
						continue;
					}
				}

				double t11, t12, t21, t22;
				//cross(site[i], b[j].site[k], t11, t12);
				//cross(site[(i - 1) % 20], b[j].site[k], t21, t22);

				//if ((t11 != DBL_MAX) || (t21 != DBL_MAX))
				//{
				//	//if ((fun == j) &&  (k == ((i - 2) % 20)))
				//	//{
				//	//	if (!(((t11 > (site[i].dist - 0.001)) && (k == ((i + 1) % 20))) || ((t21 < 0.01) && (k == ((i - 2) % 20)))))
				//	//	{
				//	//		continue;
				//	//	}
				//	//}
				//	//else
				//	//{
				//	//	continue;
				//	//}
				//	continue;
				//}

				cross(t[0], b[j].site[k], t11, t12);
				cross(t[1], b[j].site[k], t21, t22);
				cross(t[2], b[j].site[k], t12, t22);
				if ((t11 != DBL_MAX) || (t21 != DBL_MAX) || (t12 != DBL_MAX))
				{
					if (fun == j)
					{
						if (((t11 > (t[0].dist - 0.001)) && (k == ((i + 1) % 20))) || ((t21 > (t[1].dist - 0.001)) && (k == ((i + 18) % 20))))
						{
							continue;
						}
					}


					m = false;
					break;
				}
			}
			if (!m)
			{
				break;
			}
		}

		for (int k = 0; k < 20; k++)
		{
			double t11, t12, t21, t22;
			//cross(site[i], a.site[k], t11, t12);
			//cross(site[(i - 1) % 20], a.site[k], t21, t22);

			//if ((t11 != DBL_MAX) || (t21 != DBL_MAX))
			//{
			//	continue;
			//}

			cross(t[0], a.site[k], t11, t12);
			cross(t[1], a.site[k], t21, t22);
			cross(t[2], a.site[k], t12, t22);
			if ((t11 < DBL_MAX) || (t21 != DBL_MAX) || (t12 != DBL_MAX))
			{
				m = false;
				break;
			}
		}



		if (!m)
		{
			continue;
		}

		move(�ƶ�[i], i);
	}
	site.reset_seg();
}

__host__ __device__ void building::change(point ��, int index)
{
	site[index].origin = ��;
}

__host__ __device__ void building::change(point ��[20])
{
	for (int i = 0; i < 20; i++)
	{
		change(��[i], i);
	}
	site.reset_seg();
}

void building::data(std::vector<double>& ����)
{
	���� = std::vector<double>();
	����.reserve(104);
	for (int i = 0; i < 20; i++)
	{
		����.push_back(site[i].origin[0]);
		����.push_back(site[i].origin[1]);
		����.push_back(site[i].dir[0]);
		����.push_back(site[i].dir[1]);
		����.push_back(site[i].dist);
	}
	����.push_back(area());
	����.push_back(double(door[0]));
	����.push_back(double(door[1]));
	����.push_back(double(target_area));
}




//__global__ void building_move(building* ����, vector* �ƶ�, int �ߴ�)
//{
//	int i = threadIdx.x + blockIdx.x * blockDim.x;
//	if (i >= �ߴ�)
//	{
//		return;
//	}
//	����[i / 20].move(�ƶ�[i], i % 20);
//}
//
//__global__ void building_reset_seg(building* ����, int �ߴ�)
//{
//	int i = threadIdx.x + blockIdx.x * blockDim.x;
//	if (i >= �ߴ�)
//	{
//		return;
//	}
//	����[i / 20].site.reset_seg(i % 20);
//}
//
//void ��������(std::vector<building>& ����, std::vector<vector>& �ƶ�)
//{
//	int cuda�豸����;
//	hipGetDeviceCount(&cuda�豸����);
//	if (cuda�豸���� == 0)
//	{
//		for (int i = 0; i < ����.size(); i++)
//		{
//			for (int j = 0; j < �ƶ�.size(); j++)
//			{
//				����[i].move(�ƶ�[i * 20 + j], j);
//			}
//			����[i].site.reset_seg();
//		}
//	}
//	else
//	{
//		int �Կ�id;
//		hipGetDevice(&�Կ�id);
//		hipDeviceProp_t �Կ�����;
//		hipGetDeviceProperties(&�Կ�����, �Կ�id);
//		int ÿ���߳� = �Կ�����.maxThreadsPerBlock;
//		int ���� = �ƶ�.size() / ÿ���߳� + 1;
//
//		building* ����_dev = NULL;//
//		vector* �ƶ�_dev = NULL;//
//		hipMalloc((void**)����_dev, sizeof(building) * ����.size());
//		hipMalloc((void**)�ƶ�_dev, sizeof(vector) * �ƶ�.size());
//		hipMemcpy(����_dev, ����.data(), sizeof(building) * ����.size(), hipMemcpyHostToDevice);
//		hipMemcpy(�ƶ�_dev, ����.data(), sizeof(vector) * �ƶ�.size(), hipMemcpyHostToDevice);
//
//		building_move << < ����, ÿ���߳� >> > (����_dev, �ƶ�_dev, �ƶ�.size());
//
//		hipFree(�ƶ�_dev);
//
//		building_reset_seg << < ����, ÿ���߳� >> > (����_dev, �ƶ�.size());
//
//		hipMemcpy(����.data(), ����_dev, sizeof(building) * ����.size(), hipMemcpyDeviceToHost);
//		hipFree(����_dev);
//	}
//}

building ͣ��������(building �ּ���)
{
	seg ƽ��[5];
	double ��ת = 90;
	if (�ּ���.site.dir_area() > 0)
	{
		��ת = -90;
	}

	for (int i = 0; i < 5; i++)
	{
		ƽ��[i].origin = rotate(�ּ���.site[i].origin, �ּ���.site[i].point_get(27), ��ת);
		ƽ��[i].dir = �ּ���.site[i].dir;
		ƽ��[i].dist = �ּ���.site[i].dist;
	}



	point ��[20];
	for (int i = 0; i < 5; i++)
	{
		��[i] = �ּ���.site[i].origin;
	}
	��[5] = ƽ��[4].end();
	for (int i = 1; i < 4; i++)
	{
		��[i + 5] = cross(line(ƽ��[5 - i]), line(ƽ��[5 - i - 1]));
	}
	��[9] = ƽ��[0].origin;

	for (int i = 10; i < 20; i++)
	{
		��[i] = ��[0];
	}


	return building(��, 0, 4, fun_port, 0);
}



const char ������[8][8] =
{
	{0, 4, 0, 0, 0, 3, 0, 0},
	{4, 0, 4, 2, 2, 0, 0, 0},
	{0, 4, 0, 0,-1, 0, 0, 0},
	{0, 2, 0, 0,-1, 0, 0, 0},
	{0, 2,-1,-1, 0,-1,-1, 0},
	{3, 0, 0, 0,-1, 0, 3, 1},
	{0, 0, 0, 0,-1, 3, 0, 4},
	{0, 0, 0, 0, 0, 1, 4, 0}
};


double ��������(ground ����, std::vector<building>& ����, bool& reset)
{
	double ���� = 0;

	const double
		������_Ȩ�� = 0.34,
		���_Ȩ�� = 0.25,
		ƽֱ��_Ȩ�� = 0.15,
		����_Ȩ�� = 0.2,
		��_Ȩ�� = 0.2,
		�ص�_Ȩ�� = 0.33,
		�Ϸ�_Ȩ�� = 0.33,
		�ܳ�_Ȩ�� = 0.1;


	for (int i = 0; i < ����.size(); i++)
	{
		double ��� = ����[i].area();

		if (����.site.full_overlap(����[i].site))
		{
			���� += ������_Ȩ��;
		}
		else
		{
			double a = fmin(1, pow(overlap_area(����.site, ����[i].site) / ���, 2));
			���� += ������_Ȩ�� * a / 2;
			reset = true;
		}

		for (int j = 0; j < i; j++)
		{
			if (!����[i].site.is_overlap(����[j].site))
			{
				���� += �ص�_Ȩ�� / 28 * 8;
			}
			else
			{
				double a = (1 - fmin(1, pow(overlap_area(����[j].site, ����[i].site) / ���, 2)));
				���� += �ص�_Ȩ�� * a / 2 / 28 * 8;
				reset = true;
			}

			if (������[����[i].fun][����[j].fun] >= 0)
			{
				���� += exp(-dist(����[i].site, ����[j].site) * ������[����[i].fun][����[j].fun] / 100) * ����_Ȩ�� / 28 * 8;

				���� += exp(-fmin(dist(����[j].site, ����[i].get_door(0)), dist(����[j].site, ����[i].get_door(1))) * ������[����[i].fun][����[j].fun] / 100) * ��_Ȩ�� / 28 * 8;
			}
			else
			{
				���� += (1 - exp(-dist(����[i].site, ����[j].site)) / 100) * ����_Ȩ�� / 28 * 8;

				���� += (1 - exp(-fmin(dist(����[j].site, ����[i].get_door(0)), dist(����[j].site, ����[i].get_door(1))) / 100)) * ��_Ȩ�� / 28 * 8;
			}
		}

		���� += exp(-pow((��� - ����[i].target_area) / 10000, 2)) * ���_Ȩ��;

		double �ܳ� = 0;
		for (int j = 0; j < 20; j++)
		{
			double a = fmax(fmax((����[i].site[j].dir * ����[i].site[(j + 1) % 20].dir), 0), abs(����[i].site[j].dir ^ ����[i].site[(j + 1) % 20].dir));
			���� += (a + pow(a, 16)) / 2 * ƽֱ��_Ȩ�� / 20;

			�ܳ� += ����[i].site[j].dist;

			if (a < M_SQRT1_2)
			{
				reset = true;
			}
		}

		���� += fmin(1, exp(sqrt(����[i].target_area) - �ܳ�)) * �ܳ�_Ȩ��;

		if (����[i].site.legal())
		{
			���� += �Ϸ�_Ȩ��;
		}
		else
		{
			double a = fmin(1, pow(����[i].site.dir_area() / ���, 2));
			���� += �Ϸ�_Ȩ�� * a / 2;
			reset = true;
		}
	}

	return ���� / 8;
}

void �ֿ����_����(std::vector<double>& �ֿ����, std::vector<double>& ������_, std::vector<double>& ��������_, std::vector<char>& �������, std::vector<double>& �ֿ��޸�)
{
	�ֿ���� = { 0,0,0 };
	for (int i = 0; i < ������_.size(); i++)
	{
		�ֿ����[�������[i]] += (������_[i] + ��������_[i]) / �ֿ��޸�[�������[i]];
	}
}

void ����趨(std::vector<building>& ����, double ������, std::vector<double>& �ֿ����)
{
	���� = std::vector<building>(8);

	����[0].target_area = ������ / 30 / 365 / 24 * 3 * 35;
	����[1].target_area = ������ / 30 / 365 / 24 * 3 * 20;
	����[2].target_area = �ֿ����[0] * 1.7;
	����[3].target_area = �ֿ����[1] * 1.7;
	����[4].target_area = �ֿ����[2] * 1.7;
	����[5].target_area = 5000;
	����[6].target_area = ������ / 30 / 365 / 24 * (3 + 2) * 10;
	����[7].target_area = (500 * 0.7 + ������ / 30 / 365 / 24 * 3 * 0.3) * 40;

	for (int i = 0; i < ����.size(); i++)
	{
		����[i].fun = i;
	}
}
