#include "hip/hip_runtime.h"
#include "geometry.cuh"



__host__ __device__ poly::poly() {}

__host__ __device__ poly::poly(const point* ��, int m)
{
	int temp = m < 20 ? m : 20 ;
	for (int i = 0; i < temp - 1; i++)
	{
		segs[i] = seg(��[i], ��[i + 1]);
	}
	for (int i = temp; i < 20 ; i++)
	{
		segs[i] = seg(��[temp], ��[temp + 1]);
	}
	segs[20 - 1] = seg(��[temp - 1], ��[0]);
}

poly::poly(std::vector<point>& ��)
{
	int temp = (��.size() < 20 ) ? ��.size() : 20 ;
	for (int i = 0; i < temp - 1; i++)
	{
		segs[i] = seg(��[i], ��[i + 1]);
	}
	for (int i = temp - 1; i < 20 - 1; i++)
	{
		segs[i] = seg(��[��.size() - 1], ��[��.size() - 1]);
	}
	segs[20 - 1] = seg(��[temp - 1], ��[0]);
}

__host__ __device__ bool poly::legal()
{
	reset_seg();
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < 20; j++)
		{
			double t_1, t_2;
			cross(segs[i], segs[j], t_1, t_2);
			if ((t_1 != DBL_MAX) && (((abs(t_1 - segs[i].dist) > 0.0001) && (abs(t_1) > 0.0001)) || ((abs(t_2 - segs[j].dist) > 0.0001) && (abs(t_2) > 0.0001))))
			{
				return false;
			}
		}
	}
	return true;
}

__host__ __device__ void poly::point_get(point*& ��) const
{
	if (�� != nullptr)
	{
		delete[]��;
	}
	�� = new point[20];
	for (int i = 0; i < 20 ; i++)
	{
		��[i] = (segs[i]).origin;
	}
}

void poly::point_get(std::vector<point>& ��) const
{
	�� = std::vector<point>(20);
	for (int i = 0; i < 20 ; i++)
	{
		��[i] = (segs[i]).origin;
	}
}

__host__ __device__ void poly::seg_get(seg*& �߶�) const
{
	if (�߶� != nullptr)
	{
		delete[]�߶�;
	}
	�߶� = new seg[20];
	for (int i = 0; i < 20 ; i++)
	{
		�߶�[i] = (segs[i]);
	}
}

void poly::seg_get(std::vector<seg>& �߶�) const
{
	�߶� = std::vector<seg>(20);
	for (int i = 0; i < 20 ; i++)
	{
		�߶�[i] = (segs[i]);
	}
}

__host__ __device__ bool poly::point_in(point ��) const
{
	ray temp;
	temp.origin = ��;
	temp.dir = vector(point({ 0,1 }));
	int k = 0;

	point max = segs[0].origin, min = segs[0].origin;
	for (int i = 1; i < 20 ; i++)
	{
		max[0] = (max[0] > segs[i].origin[0]) ? max[0] : segs[i].origin[0];
		max[1] = (max[1] > segs[i].origin[0]) ? max[1] : segs[i].origin[1];
		min[0] = (min[0] < segs[i].origin[0]) ? min[0] : segs[i].origin[0];
		min[1] = (min[1] < segs[i].origin[0]) ? min[1] : segs[i].origin[1];
	}
	if ((max[0] < ��[0]) || (max[1] < ��[1]) || (min[0] > ��[0]) || (min[1] > ��[1]))
	{
		return false;
	}

	for (int i = 0; i < 20 ; i++)
	{
		if (is_cross(temp, segs[i]))
		{
			k++;
		}
	}
	if ((k % 2) == 0)
	{
		return false;
	}

	temp.dir = vector(point({ 0,-1 }));
	k = 0;
	for (int i = 0; i < 20 ; i++)
	{
		if (is_cross(temp, segs[i]))
		{
			k++;
		}
	}
	if ((k % 2) == 0)
	{
		return false;
	}
	return true;
}

__host__ __device__ void poly::reset_seg()
{
	for (int i = 0, n = 0; (i < 20 - 1) && (n < 20); i++)
	{
		if ((abs(segs[i].origin[0] - segs[i + 1].origin[0]) > 0.001) || (abs(segs[i].origin[1] - segs[i + 1].origin[1]) > 0.001))
		{
			continue;
		}
		n++;
		i--;
		for (int j = i + 1; j < 20 - 1; j++)
		{
			segs[j].origin = segs[j + 1].origin;
		}
		segs[19].origin = segs[0].origin;
	}
	


	for (int i = 0; i < 20 - 1; i++)
	{
		segs[i] = seg(segs[i].origin, segs[i + 1].origin);
	}
	segs[19] = seg(segs[19].origin, segs[0].origin);
}

__host__ __device__ void poly::reset_seg(int i)
{
	segs[i] = seg(segs[i].origin, segs[(i + 1) % 20].origin);
}

__host__ __device__ seg& poly::operator[](int i)
{
	return segs[i % 20 ];
}

__host__ __device__ seg poly::operator[](int i) const
{
	return segs[i % 20 ];
}

__host__ __device__ double poly::area() const
{
	double s = 0;
	for (int i = 0; i < 20 - 1; i++)
	{
		s += vector(segs[i].origin) ^ vector(segs[i + 1].origin);
	}
	s += vector(segs[20 - 1].origin) ^ vector(segs[0].origin);
	return s / 2;
}

void poly::print(cv::InputOutputArray ͼ��, double ����, const cv::Scalar& ��ɫ, int ��ϸ) const
{
	//seg(segs[0].origin, segs[1].origin).print(ͼ��, ����, ��ɫ, ��ϸ * 2);
	for (int i = 0; i < 19; i++)
	{
		seg(segs[i].origin, segs[i + 1].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
	}
	seg(segs[19].origin, segs[0].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
	//segs[0].origin.print(ͼ��, ����, ��ɫ, ��ϸ * 4);
}

vector poly::move2center()
{
	reset_seg();


	double s = area();
	double x = 0, y = 0;
	for (int i = 0; i < 19; i++)
	{
		double �� = segs[i].dir ^ segs[i + 1].dir;
		x += (segs[i].origin[0] + segs[i+1].origin[0]) * ��;
		y += (segs[i].origin[1] + segs[i+1].origin[1]) * ��;
	}
	vector move(x / 6 / s, y / 6 / s);

	//for (int i = 0; i < 19; i++)
	//{
	//	if ((abs(segs[i].origin[0] - segs[i + 1].origin[0]) > 0.00001) || (abs(segs[i].origin[1] - segs[i + 1].origin[1]) > 0.00001))
	//	{
	//		move -= vector(segs[i].origin);
	//		n++;
	//	}
	//}
	//if ((abs(segs[19].origin[0] - segs[0].origin[0]) > 0.00001) || (abs(segs[19].origin[1] - segs[0].origin[1]) > 0.00001))
	//{
	//	move -= vector(segs[19].origin);
	//	n++;
	//}
	//
	//move /= n;
	for (int i = 0; i < 20; i++)
	{
		segs[i].origin = point(vector(segs[i].origin) + move);
	}

	return move;
}

__host__ __device__ void poly::simple(double �Ƕ�, bool rad)
{
	if (!rad)
	{
		�Ƕ� = deg2rad(�Ƕ�);
	}
	double cos_ = cos(�Ƕ�);

	reset_seg();
	int n = 1;
	while (n != 0)	{

		n = 0;
		for (int i = 0, j = 1; j < 20; j++)
		{
			i = j - 1;

			double cos_t = (vector(0.0, 0.0) - segs[i].dir) * segs[j].dir;
			if ((cos_t > cos_) && (segs[i].dist > 0.0001) && (segs[j].dist > 0.0001))
			{
				n++;
				if (i == 18)
				{
					segs[19].origin = segs[0].origin;
				}
				for (int k = i + 1; k < 20 - 1; k++)
				{
					segs[k].origin = segs[k + 1].origin;
				}
				reset_seg();
			}
		}
		vector dir_;
		for (int i = 19; i >= 0; i--)
		{
			if (segs[i].dist > 0.0001)
			{
				dir_ = segs[i].dir;
				break;
			}
		}
		double cos_t = (vector(0.0, 0.0) - dir_) * segs[0].dir;
		if (cos_t > cos_)
		{
			n++;
			for (int j = 0; j < 20 - 1; j++)
			{
				segs[j].origin = segs[j + 1].origin;
			}
			reset_seg();
		}
	}
}

__host__ __device__ bool poly::is_overlap(const poly other) const
{
	return ::is_overlap(*this, other);
}

__host__ __device__ bool poly::full_overlap(const poly other) const
{
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < 20; j++)
		{
			if (is_cross(other[i], segs[j]))
			{
				return false;
			}
		}
	}
	if (!point_in(other[0].origin))
	{
		return false;
	}
	return true;
}

__host__ __device__ bool is_overlap(const poly p_1, const poly p_2)
{
	int l[20];
	for (int i = 0; i < 20; i++)
	{
		l[i] = 0;
	}

	for (int i = 0; i < 20 ; i++)
	{
		int k = 0;
		for (int j = 0; j < 20; j++)
		{
			double t_1, t_2;
			cross(ray(p_1[i]), ray(p_2[j]), t_1, t_2);
			if ((t_1 < p_1[i].dist) && (t_2 < p_2[j].dist))
			{
				return true;
			}

			if ((t_1 != DBL_MAX) || ((t_2 > p_2[j].dist) && (t_2 != DBL_MAX)))
			{
				l[j]++;
				k++;
			}
		}

		if ((k % 2) == 0)
		{
			continue;
		}

		k = 0;
		for (int j = 0; j < 20; j++)
		{
			double t_1, t_2;
			cross(ray(p_1[i].origin, -1 * p_1[i].dir), p_2[j], t_1, t_2);

			if (t_1 != DBL_MAX)
			{
				k++;
			}
		}

		if ((k % 2) == 1)
		{
			return true;
		}
	}

	for (int i = 0; i < 20; i++)
	{
		if ((l[i] % 2) == 0)
		{
			continue;
		}

		l[i] = 0;
		for (int j = 0; j < 20 ; j++)
		{
			double t_1, t_2;
			cross(p_1[j], ray(p_2[i].origin, -1 * p_2[i].dir), t_1, t_2);

			if (t_1 != DBL_MAX)
			{
				l[i]++;
			}
		}

		if ((l[i] % 2) == 1)
		{
			return true;
		}
	}

	return false;
}

__host__ __device__ double dist(const poly p_1, const poly p_2)
{
	double dist = DBL_MAX;
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < 20; j++)
		{
			dist = fmin(dist, p_1[i].point_dist(p_2[j].origin));
			dist = fmin(dist, p_2[i].point_dist(p_1[j].origin));
		}
	}
	return dist;
}
