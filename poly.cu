#include "hip/hip_runtime.h"
#include "geometry.cuh"



__host__ __device__ poly::poly() {}

__host__ __device__ poly::poly(const point* ��, int m)
{
	int temp = m < 20 ? m : 20 ;
	for (int i = 0; i < temp - 1; i++)
	{
		segs[i] = seg(��[i], ��[i + 1]);
	}
	for (int i = temp; i < 20 ; i++)
	{
		segs[i] = seg(��[temp], ��[temp + 1]);
	}
	segs[20 - 1] = seg(��[temp - 1], ��[0]);
}

poly::poly(std::vector<point>& ��)
{
	int temp = (��.size() < 20 ) ? ��.size() : 20 ;
	for (int i = 0; i < temp - 1; i++)
	{
		segs[i] = seg(��[i], ��[i + 1]);
	}
	for (int i = temp - 1; i < 20 - 1; i++)
	{
		segs[i] = seg(��[��.size() - 1], ��[��.size() - 1]);
	}
	segs[20 - 1] = seg(��[temp - 1], ��[0]);
}

__host__ __device__ bool poly::legal()
{
	reset_seg();
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < i; j++)
		{
			double t_1, t_2;
			cross(segs[i], segs[j], t_1, t_2);
			if ((t_1 != DBL_MAX) && (((abs(t_1 - segs[i].dist) > 0.01) && (abs(t_1) > 0.01)) || ((abs(t_2 - segs[j].dist) > 0.01) && (abs(t_2) > 0.01))))
			{
				return false;
			}
		}
	}
	return true;
}

__host__ __device__ void poly::point_get(point*& ��) const
{
	if (�� != nullptr)
	{
		delete[]��;
	}
	�� = new point[20];
	for (int i = 0; i < 20 ; i++)
	{
		��[i] = (segs[i]).origin;
	}
}

void poly::point_get(std::vector<point>& ��) const
{
	�� = std::vector<point>(20);
	for (int i = 0; i < 20 ; i++)
	{
		��[i] = (segs[i]).origin;
	}
}

__host__ __device__ void poly::seg_get(seg*& �߶�) const
{
	if (�߶� != nullptr)
	{
		delete[]�߶�;
	}
	�߶� = new seg[20];
	for (int i = 0; i < 20 ; i++)
	{
		�߶�[i] = (segs[i]);
	}
}

void poly::seg_get(std::vector<seg>& �߶�) const
{
	�߶� = std::vector<seg>(20);
	for (int i = 0; i < 20 ; i++)
	{
		�߶�[i] = (segs[i]);
	}
}

__host__ __device__ bool poly::point_in(point ��) const
{
	ray temp;
	temp.origin = ��;
	temp.dir = vector(point({ 0,1 }));
	int k = 0;

	point max = segs[0].origin, min = segs[0].origin;
	for (int i = 1; i < 20 ; i++)
	{
		max[0] = (max[0] > segs[i].origin[0]) ? max[0] : segs[i].origin[0];
		max[1] = (max[1] > segs[i].origin[0]) ? max[1] : segs[i].origin[1];
		min[0] = (min[0] < segs[i].origin[0]) ? min[0] : segs[i].origin[0];
		min[1] = (min[1] < segs[i].origin[0]) ? min[1] : segs[i].origin[1];
	}
	if ((max[0] < ��[0]) || (max[1] < ��[1]) || (min[0] > ��[0]) || (min[1] > ��[1]))
	{
		return false;
	}

	for (int i = 0; i < 20 ; i++)
	{
		if (is_cross(temp, segs[i]))
		{
			k++;
		}
	}
	if ((k % 2) == 0)
	{
		return false;
	}

	temp.dir = vector(point({ 0,-1 }));
	k = 0;
	for (int i = 0; i < 20 ; i++)
	{
		if (is_cross(temp, segs[i]))
		{
			k++;
		}
	}
	if ((k % 2) == 0)
	{
		return false;
	}
	return true;
}

__host__ __device__ void poly::reset_seg()
{
	for (int i = 0; i < 20 - 1; i++)
	{
		segs[i] = seg(segs[i].origin, segs[i + 1].origin);
	}
	segs[19] = seg(segs[20 - 1].origin, segs[0].origin);
}

__host__ __device__ void poly::reset_seg(int i)
{
	segs[i] = seg(segs[i].origin, segs[(i + 1) % 20].origin);
}

__host__ __device__ seg& poly::operator[](int i)
{
	return segs[i % 20 ];
}

__host__ __device__ seg poly::operator[](int i) const
{
	return segs[i % 20 ];
}

__host__ __device__ double poly::area() const
{
	double s = 0;
	for (int i = 0; i < 20 - 1; i++)
	{
		s += vector(segs[i].origin) ^ vector(segs[i + 1].origin);
	}
	s += vector(segs[20 - 1].origin) ^ vector(segs[0].origin);
	return s / 2;
}

void poly::print(cv::InputOutputArray ͼ��, double ����, const cv::Scalar& ��ɫ, int ��ϸ) const
{
	for (int i = 0; i < 19; i++)
	{
		seg(segs[i].origin, segs[i + 1].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
	}
	seg(segs[19].origin, segs[0].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
}

vector poly::move2center()
{
	int n = 0;
	vector move(0.0, 0.0);
	for (int i = 0; i < 19; i++)
	{
		if ((abs(segs[i].origin[0] - segs[19].origin[0]) > 0.00001) || (abs(segs[i].origin[1] - segs[19].origin[1]) > 0.00001))
		{
			move -= vector(segs[i].origin);
			n++;
		}
	}
	move -= vector(segs[19].origin);
	n++;

	move /= n;
	for (int i = 0; i < 20; i++)
	{
		segs[i].origin = point(vector(segs[i].origin) + move);
	}

	return move;
}

__host__ __device__ bool poly::is_overlap(const poly other) const
{
	return ::is_overlap(*this, other);
}

__host__ __device__ bool poly::full_overlap(const poly other) const
{
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < 20; j++)
		{
			if (is_cross(other[i], segs[j]))
			{
				return false;
			}
		}
	}
	if (!point_in(other[0].origin))
	{
		return false;
	}
	return true;
}

__host__ __device__ bool is_overlap(const poly p_1, const poly p_2)
{
	int l[20];
	for (int i = 0; i < 20; i++)
	{
		l[i] = 0;
	}

	for (int i = 0; i < 20 ; i++)
	{
		int k = 0;
		for (int j = 0; j < 20; j++)
		{
			double t_1, t_2;
			cross(ray(p_1[i]), ray(p_2[j]), t_1, t_2);
			if ((t_1 < p_1[i].dist) && (t_2 < p_2[j].dist))
			{
				return true;
			}

			if ((t_1 != DBL_MAX) || ((t_2 > p_2[j].dist) && (t_2 != DBL_MAX)))
			{
				l[j]++;
				k++;
			}
		}

		if ((k % 2) == 0)
		{
			continue;
		}

		k = 0;
		for (int j = 0; j < 20; j++)
		{
			double t_1, t_2;
			cross(ray(p_1[i].origin, -1 * p_1[i].dir), p_2[j], t_1, t_2);

			if (t_1 != DBL_MAX)
			{
				k++;
			}
		}

		if ((k % 2) == 1)
		{
			return true;
		}
	}

	for (int i = 0; i < 20; i++)
	{
		if ((l[i] % 2) == 0)
		{
			continue;
		}

		l[i] = 0;
		for (int j = 0; j < 20 ; j++)
		{
			double t_1, t_2;
			cross(p_1[j], ray(p_2[i].origin, -1 * p_2[i].dir), t_1, t_2);

			if (t_1 != DBL_MAX)
			{
				l[i]++;
			}
		}

		if ((l[i] % 2) == 1)
		{
			return true;
		}
	}

	return false;
}

__host__ __device__ double dist(const poly p_1, const poly p_2)
{
	double dist = DBL_MAX;
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < 20; j++)
		{
			dist = fmin(dist, p_1[i].point_dist(p_2[j].origin));
			dist = fmin(dist, p_2[i].point_dist(p_1[j].origin));
		}
	}
	return dist;
}
