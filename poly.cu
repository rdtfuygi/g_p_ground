#include "hip/hip_runtime.h"
#include "geometry.cuh"



__host__ __device__ poly::poly() {}

__host__ __device__ poly::poly(const point* ��, int m)
{
	int temp = m < 20 ? m : 20 ;
	for (int i = 0; i < temp - 1; i++)
	{
		segs[i] = seg(��[i], ��[i + 1]);
	}
	for (int i = temp; i < 20 ; i++)
	{
		segs[i] = seg(��[temp], ��[temp + 1]);
	}
	segs[20 - 1] = seg(��[temp - 1], ��[0]);
}

poly::poly(std::vector<point>& ��)
{
	int temp = (��.size() < 20 ) ? ��.size() : 20 ;
	for (int i = 0; i < temp - 1; i++)
	{
		segs[i] = seg(��[i], ��[i + 1]);
	}
	for (int i = temp - 1; i < 20 - 1; i++)
	{
		segs[i] = seg(��[��.size() - 1], ��[��.size() - 1]);
	}
	segs[20 - 1] = seg(��[temp - 1], ��[0]);
}

__host__ __device__ bool poly::legal()
{
	reset_seg();
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < 20; j++)
		{
			double t_1, t_2;
			cross(segs[i], segs[j], t_1, t_2);
			if ((t_1 != DBL_MAX) && (((abs(t_1 - segs[i].dist) > 0.0001) && (abs(t_1) > 0.0001)) || ((abs(t_2 - segs[j].dist) > 0.0001) && (abs(t_2) > 0.0001))))
			{
				return false;
			}
		}
	}
	return true;
}

__host__ __device__ double poly::one_link_area()
{
	point max = segs[0].origin, min = segs[0].origin;
	for (int i = 1; i < 20; i++)
	{
		max[0] = (max[0] > segs[i].origin[0]) ? max[0] : segs[i].origin[0];
		max[1] = (max[1] > segs[i].origin[1]) ? max[1] : segs[i].origin[1];
		min[0] = (min[0] < segs[i].origin[0]) ? min[0] : segs[i].origin[0];
		min[1] = (min[1] < segs[i].origin[1]) ? min[1] : segs[i].origin[1];
	}


	double last[20];
	{
		ray temp;
		temp.origin = point(int(min[0] + 1), min[1]);
		temp.dir = vector(0.0, 1.0);
		for (int i = 0; i < 20; i++)
		{
			double t_1, t_2;
			cross(temp, segs[i], t_1, t_2);
			last[i] = t_1;
		}
		for (int i = 19; i > 0; i--)
		{
			bool swap = false;
			for (int j = 0; j < i; j++)
			{
				if (last[j] < last[j + 1])
				{
					continue;
				}
				double temp_dist = last[j];
				last[j] = last[j + 1];
				last[j + 1] = temp_dist;
				swap = true;
			}
			if (!swap)
			{
				break;
			}
		}
	}


	double areas[10];
	char map[10] = { 0,1,2,3,4,5,6,7,8,9 };
	for (int i = 0; i < 10; i++)
	{
		if ((last[2 * i + 1] != DBL_MAX) && (last[2 * i] != DBL_MAX))
		{
			areas[i] = last[2 * i + 1] - last[2 * i];
		}
		else
		{
			areas[i] = 0;
		}
	}

	for (int x = min[0] + 2; x < max[0]; x++)
	{
		double dist[20];
		char map_new[10] = { 10,10,10,10,10,10,10,10,10,10 };

		seg temp;
		temp.origin = point(x, min[1]);
		temp.dir = vector(0.0, 1.0);
		temp.dist = max[1] - min[1];
		for (int i = 0; i < 20; i++)
		{
			double t_1, t_2;
			cross(temp, segs[i], t_1, t_2);
			dist[i] = t_1;
		}
		for (int i = 19; i > 0; i--)
		{
			bool swap = false;
			for (int j = 0; j < i; j++)
			{
				if (dist[j] > dist[j + 1])
				{
					double temp_dist = dist[j];
					dist[j] = dist[j + 1];
					dist[j + 1] = temp_dist;
					swap = true;
				}
			}
			if (!swap)
			{
				break;
			}
		}

		int i = 0, j = 0;
		while ((i < 10) && (j < 10))
		{
			if ((last[2 * i] == DBL_MAX) || (last[2 * i + 1] == DBL_MAX) || (dist[2 * j] == DBL_MAX) || (dist[2 * j + 1] == DBL_MAX))
			{
				break;
			}
			if ((last[2 * i + 1] > dist[2 * j]) && (last[2 * i] < dist[2 * j + 1]))
			{
				if (map_new[j] == 10)
				{
					map_new[j] = map[i];
					areas[map_new[j]] += dist[2 * j + 1] - dist[2 * j];
				}
				else if (map_new[j] != map[i])
				{
					areas[map_new[j]] += areas[map[i]];
				}

			}
			if (last[2 * i + 1] < dist[2 * j + 1])
			{
				i++;
			}
			else if (last[2 * i + 1] > dist[2 * j + 1])
			{
				j++;
			}
			else
			{
				i++;
				j++;
			}
		}
		for (int i = 0; i < 10; i++)
		{
			last[2 * i] = dist[2 * i];
			last[2 * i + 1] = dist[2 * i + 1];
			if (map_new[i] != 10)
			{
				map[i] = map_new[i];
			}
		}
	}
	double output = 0;
	for (int i = 0; i < 10; i++)
	{
		output = (areas[i] > output) ? areas[i] : output;
	}

	return output;
}

__host__ __device__ void poly::point_get(point*& ��) const
{
	if (�� != nullptr)
	{
		delete[]��;
	}
	�� = new point[20];
	for (int i = 0; i < 20 ; i++)
	{
		��[i] = (segs[i]).origin;
	}
}

void poly::point_get(std::vector<point>& ��) const
{
	�� = std::vector<point>(20);
	for (int i = 0; i < 20 ; i++)
	{
		��[i] = (segs[i]).origin;
	}
}

__host__ __device__ void poly::seg_get(seg*& �߶�) const
{
	if (�߶� != nullptr)
	{
		delete[]�߶�;
	}
	�߶� = new seg[20];
	for (int i = 0; i < 20 ; i++)
	{
		�߶�[i] = (segs[i]);
	}
}

void poly::seg_get(std::vector<seg>& �߶�) const
{
	�߶� = std::vector<seg>(20);
	for (int i = 0; i < 20 ; i++)
	{
		�߶�[i] = (segs[i]);
	}
}

__host__ __device__ bool poly::point_in(point ��) const
{
	ray temp;
	temp.origin = ��;
	temp.dir = vector(point({ 0,1 }));
	int k = 0;

	point max = segs[0].origin, min = segs[0].origin;
	for (int i = 1; i < 20 ; i++)
	{
		max[0] = (max[0] > segs[i].origin[0]) ? max[0] : segs[i].origin[0];
		max[1] = (max[1] > segs[i].origin[1]) ? max[1] : segs[i].origin[1];
		min[0] = (min[0] < segs[i].origin[0]) ? min[0] : segs[i].origin[0];
		min[1] = (min[1] < segs[i].origin[1]) ? min[1] : segs[i].origin[1];
	}
	if ((max[0] < ��[0]) || (max[1] < ��[1]) || (min[0] > ��[0]) || (min[1] > ��[1]))
	{
		return false;
	}

	for (int i = 0; i < 20 ; i++)
	{
		if (is_cross(temp, segs[i]))
		{
			k++;
		}
	}
	if ((k % 2) == 0)
	{
		return false;
	}

	temp.dir = vector(point({ 0,-1 }));
	k = 0;
	for (int i = 0; i < 20 ; i++)
	{
		if (is_cross(temp, segs[i]))
		{
			k++;
		}
	}
	if ((k % 2) == 0)
	{
		return false;
	}
	return true;
}

__host__ __device__ void poly::reset_seg()
{
	for (int i = 0, n = 0; (i < 20 - 1) && (n < 20); i++)
	{
		if ((abs(segs[i].origin[0] - segs[i + 1].origin[0]) > 0.001) || (abs(segs[i].origin[1] - segs[i + 1].origin[1]) > 0.001))
		{
			continue;
		}
		n++;
		i--;
		for (int j = i + 1; j < 20 - 1; j++)
		{
			segs[j].origin = segs[j + 1].origin;
		}
		segs[19].origin = segs[0].origin;
	}
	


	for (int i = 0; i < 20 - 1; i++)
	{
		segs[i] = seg(segs[i].origin, segs[i + 1].origin);
	}
	segs[19] = seg(segs[19].origin, segs[0].origin);
}

__host__ __device__ void poly::reset_seg(int i)
{
	segs[i] = seg(segs[i].origin, segs[(i + 1) % 20].origin);
}

__host__ __device__ seg& poly::operator[](int i)
{
	return segs[i % 20 ];
}

__host__ __device__ seg poly::operator[](int i) const
{
	return segs[i % 20 ];
}

__host__ __device__ double poly::dir_area() const
{
	double s = 0;
	for (int i = 0; i < 20 - 1; i++)
	{
		s += vector(segs[i].origin) ^ vector(segs[i + 1].origin);
	}
	s += vector(segs[20 - 1].origin) ^ vector(segs[0].origin);
	return s / 2;
}

__host__ __device__ double poly::area() const
{
	point max = segs[0].origin, min = segs[0].origin;
	for (int i = 1; i < 20; i++)
	{
		max[0] = (max[0] > segs[i].origin[0]) ? max[0] : segs[i].origin[0];
		max[1] = (max[1] > segs[i].origin[1]) ? max[1] : segs[i].origin[1];
		min[0] = (min[0] < segs[i].origin[0]) ? min[0] : segs[i].origin[0];
		min[1] = (min[1] < segs[i].origin[1]) ? min[1] : segs[i].origin[1];
	}
	double output = 0;
	for (int x = min[0]; x < max[0]; x++)
	{
		seg temp;
		temp.origin = point(x, min[1]);
		temp.dir = vector(0.0, 1.0);
		temp.dist = max[1] - min[1];

		double dist[20];

		for (int i = 0; i < 20; i++)
		{
			double t_1, t_2;
			cross(temp, segs[i], t_1, t_2);
			dist[i] = t_1;
		}
		for (int i = 19; i > 0; i--)
		{
			bool swap = false;
			for (int j = 0; j < i; j++)
			{
				if (dist[j] > dist[j + 1])
				{
					double temp_dist = dist[j];
					dist[j] = dist[j + 1];
					dist[j + 1] = temp_dist;
					swap = true;
				}
			}
			if (!swap)
			{
				break;
			}
		}

		for (int i = 0; i < 10; i++)
		{
			if ((dist[2 * i + 1] == DBL_MAX) || (dist[2 * i] == DBL_MAX))
			{
				break;
			}
			output += dist[2 * i + 1] - dist[2 * i];
		}
	}
	return output;
}

void poly::print(cv::InputOutputArray ͼ��, double ����, const cv::Scalar& ��ɫ, int ��ϸ) const
{
	//seg(segs[0].origin, segs[1].origin).print(ͼ��, ����, ��ɫ, ��ϸ * 2);
	for (int i = 0; i < 19; i++)
	{
		seg(segs[i].origin, segs[i + 1].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
	}
	seg(segs[19].origin, segs[0].origin).print(ͼ��, ����, ��ɫ, ��ϸ);
	//segs[0].origin.print(ͼ��, ����, ��ɫ, ��ϸ * 4);
}

vector poly::move2center()
{
	reset_seg();


	double s = (dir_area() > 0) ? area() : -area();
	double x = 0, y = 0;
	for (int i = 0; i < 19; i++)
	{
		double �� = segs[i].dir ^ segs[i + 1].dir;
		x += (segs[i].origin[0] + segs[i+1].origin[0]) * ��;
		y += (segs[i].origin[1] + segs[i+1].origin[1]) * ��;
	}
	vector move(x / 6 / s, y / 6 / s);

	for (int i = 0; i < 20; i++)
	{
		segs[i].origin = point(vector(segs[i].origin) + move);
	}

	return move;
}

__host__ __device__ void poly::simple(double �Ƕ�, bool rad)
{
	if (!rad)
	{
		�Ƕ� = deg2rad(�Ƕ�);
	}
	double cos_ = cos(�Ƕ�);

	reset_seg();
	int n = 1;
	while (n != 0)	{

		n = 0;
		for (int i = 0, j = 1; j < 20; j++)
		{
			i = j - 1;

			double cos_t = (vector(0.0, 0.0) - segs[i].dir) * segs[j].dir;
			if ((cos_t < cos_) || (segs[i].dist < 0.0001) || (segs[j].dist < 0.0001))
			{
				continue;
			}

			n++;
			if (i == 18)
			{
				segs[19].origin = segs[0].origin;
			}

			for (int k = i + 1; k < 20 - 1; k++)
			{
				segs[k].origin = segs[k + 1].origin;
			}
			reset_seg();
		}

		vector dir_;
		for (int i = 19; i >= 0; i--)
		{
			if (segs[i].dist > 0.0001)
			{
				dir_ = segs[i].dir;
				break;
			}
		}

		double cos_t = (vector(0.0, 0.0) - dir_) * segs[0].dir;
		if (cos_t < cos_)
		{
			continue;
		}

		n++;
		for (int j = 0; j < 20 - 1; j++)
		{
			segs[j].origin = segs[j + 1].origin;
		}
		reset_seg();
	}
}

__host__ __device__ bool poly::is_overlap(const poly other) const
{
	return ::is_overlap(*this, other);
}

__host__ __device__ bool poly::full_overlap(const poly other) const
{
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < 20; j++)
		{
			if (is_cross(other[i], segs[j]))
			{
				return false;
			}
		}
		if (!point_in(other[i].origin))
		{
			return false;
		}
	}
	return true;
}

__host__ __device__ double poly::overlap_area(const poly other) const
{
	return ::overlap_area(*this, other);
}

__host__ __device__ bool is_overlap(const poly p_1, const poly p_2)
{
	int l[20];
	for (int i = 0; i < 20; i++)
	{
		l[i] = 0;
	}

	for (int i = 0; i < 20 ; i++)
	{
		int k = 0;
		for (int j = 0; j < 20; j++)
		{
			double t_1, t_2;
			cross(ray(p_1[i]), ray(p_2[j]), t_1, t_2);
			if ((t_1 < p_1[i].dist) && (t_2 < p_2[j].dist))
			{
				return true;
			}

			if ((t_1 != DBL_MAX) || ((t_2 > p_2[j].dist) && (t_2 != DBL_MAX)))
			{
				l[j]++;
				k++;
			}
		}

		if ((k % 2) == 0)
		{
			continue;
		}
		
		k = 0;
		for (int j = 0; j < 20; j++)
		{
			double t_1, t_2;
			cross(ray(p_1[i].origin, -1 * p_1[i].dir), p_2[j], t_1, t_2);
		
			if (t_1 != DBL_MAX)
			{
				k++;
			}
		}

		if ((k % 2) == 1)
		{
			return true;
		}
	}

	for (int i = 0; i < 20; i++)
	{
		if ((l[i] % 2) == 0)
		{
			continue;
		}
		
		l[i] = 0;
		for (int j = 0; j < 20 ; j++)
		{
			double t_1, t_2;
			cross(p_1[j], ray(p_2[i].origin, -1 * p_2[i].dir), t_1, t_2);
		
			if (t_1 != DBL_MAX)
			{
				l[i]++;
			}
		}

		if ((l[i] % 2) == 1)
		{
			return true;
		}
	}

	return false;
}

__host__ __device__ double overlap_area(const poly p_1, const poly p_2)
{
	point max_1 = p_1.segs[0].origin, min_1 = p_1.segs[0].origin;
	for (int i = 1; i < 20; i++)
	{
		max_1[0] = (max_1[0] > p_1.segs[i].origin[0]) ? max_1[0] : p_1.segs[i].origin[0];
		max_1[1] = (max_1[1] > p_1.segs[i].origin[1]) ? max_1[1] : p_1.segs[i].origin[1];
		min_1[0] = (min_1[0] < p_1.segs[i].origin[0]) ? min_1[0] : p_1.segs[i].origin[0];
		min_1[1] = (min_1[1] < p_1.segs[i].origin[1]) ? min_1[1] : p_1.segs[i].origin[1];
	}
	point max_2 = p_2.segs[0].origin, min_2 = p_2.segs[0].origin;
	for (int i = 0; i < 20; i++)
	{
		max_2[0] = (max_2[0] > p_2.segs[i].origin[0]) ? max_2[0] : p_2.segs[i].origin[0];
		max_2[1] = (max_2[1] > p_2.segs[i].origin[1]) ? max_2[1] : p_2.segs[i].origin[1];
		min_2[0] = (min_2[0] < p_2.segs[i].origin[0]) ? min_2[0] : p_2.segs[i].origin[0];
		min_2[1] = (min_2[1] < p_2.segs[i].origin[1]) ? min_2[1] : p_2.segs[i].origin[1];
	}

	point max(fmin(max_1[0], max_2[0]), fmin(max_1[1], max_2[1])), min(fmax(min_1[0], min_2[0]), fmax(min_1[1], min_2[1]));

	double output = 0;
	for (int i = min[0]; i < max[0]; i++)
	{
		ray temp;
		temp.origin = point(i, min[1]);
		temp.dir = vector(0.0, 1.0);

		
		bool in_1 = false, in_2 = false;
		double dist[2][20];
		for (int j = 0; j < 20; j++)
		{
			double t_1, t_2;
			cross(temp, p_1.segs[j], t_1, t_2);
			dist[0][j] = t_1;
			if (t_1 != DBL_MAX)
			{
				in_1 = !in_1;
			}
			cross(temp, p_2.segs[j], t_1, t_2);
			dist[1][j] = t_1;
			if (t_1 != DBL_MAX)
			{
				in_2 = !in_2;
			}
		}
		for (int j = 19; j > 0; j--)
		{
			bool swap = false;
			for (int k = 0; k < j; k++)
			{
				if (dist[0][k] > dist[0][k + 1])
				{
					swap = true;
					double t = dist[0][k];
					dist[0][k + 1] = dist[0][k];
					dist[0][k] = t;
				}
			}
		}
		for (int j = 19; j > 0; j--)
		{
			bool swap = false;
			for (int k = 0; k < j; k++)
			{
				if (dist[1][k] > dist[1][k + 1])
				{
					swap = true;
					double t = dist[1][k];
					dist[1][k + 1] = dist[1][k];
					dist[1][k] = t;
				}
			}
		}

		int j = 0, k = 0;
		while ((j < 20) && (k < 20))
		{
			double next_1 = min[1] + dist[0][j] - temp.origin[1], next_2 = min[1] + dist[1][k] - temp.origin[1];

			if (in_1 && in_2 && ((next_1 != DBL_MAX) || (next_2 != DBL_MAX)))
			{
				output += fmin(next_1, next_2);
			}
			if (next_1 < next_2)
			{
				j++;
				in_1 = !in_1;
			}
			else if (next_1 > next_2)
			{
				k++;
				in_2 = !in_2;
			}
			else
			{
				j++;
				k++;
				in_1 = !in_1;
				in_2 = !in_2;
			}

		}
	}
	return output;
}

__host__ __device__ double dist(const poly p_1, const poly p_2)
{
	double dist = DBL_MAX;
	for (int i = 0; i < 20; i++)
	{
		for (int j = 0; j < 20; j++)
		{
			dist = fmin(dist, p_1[i].point_dist(p_2[j].origin));
			dist = fmin(dist, p_2[i].point_dist(p_1[j].origin));
		}
	}
	return dist;
}
