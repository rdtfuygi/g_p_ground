#include "hip/hip_runtime.h"
#include "geometry.cuh"
#define _USE_MATH_DEFINES 
#include <math.h>

__host__ __device__ vector::vector() :point(1, 0) {}

__host__ __device__ vector::vector(double x, double y) : point(x, y) {}

__host__ __device__ vector::vector(point ��) : point(��) {};

__host__ __device__ vector::vector(double ����[2], double ����)
{
	double ���� = ���� / ::length({ 0,0 }, ����);
	locat[0] = ����[0] * ����;
	locat[1] = ����[1] * ����;
}

__host__ __device__ vector::vector(double �Ƕ�, bool rad, double ����)
{
	if (!rad)
	{
		�Ƕ� = deg2rad(�Ƕ�);
	}
	locat[0] = cos(�Ƕ�) * ����;
	locat[1] = sin(�Ƕ�) * ����;
}

__host__ __device__ vector& vector::operator+=(vector ����)
{
	locat[0] += ����[0];
	locat[1] += ����[1];
	return *this;
}

__host__ __device__ vector& vector::operator-=(vector ����)
{
	locat[0] -= ����[0];
	locat[1] -= ����[1];
	return *this;
}

__host__ __device__ vector& vector::operator*=(double ��)
{
	locat[0] *= ��;
	locat[1] *= ��;
	return *this;
}

__host__ __device__ vector& vector::operator/=(double ��)
{
	locat[0] /= ��;
	locat[1] /= ��;
	return *this;
}

__host__ __device__ vector vector::unitize() const
{
	double ���� = length();
	if (���� < 1e-16)
	{
		return vector(M_SQRT1_2, M_SQRT1_2);
	}
	return vector(*this / ����);
}

__host__ __device__ double vector::length() const
{
	return ::length(*this);
}

__host__ __device__ vector vector::rotate(double �Ƕ�, bool rad) const
{
	return vector(::rotate({ 0,0 }, point(*this), �Ƕ�, rad));
}

__host__ __device__ double vector::angle_get(bool rad) const
{
	double �Ƕ� = atan(locat[1] / locat[0]) + (locat[0] > 0 ? 0 : M_PI);
	if (!rad)
	{
		�Ƕ� = rad2deg(�Ƕ�);
	}
	return �Ƕ�;
}

void vector::print(cv::InputOutputArray ͼ��, double ����, const cv::Scalar& ��ɫ, int ��ϸ) const
{
	int �� = ͼ��.rows(), �� = ͼ��.cols();
	int ԭ��_x = �� / 2, ԭ��_y = �� / 2;

	int �Ŵ� = 2 * (�� > �� ? �� : ��);

	cv::Point ��_1(ԭ��_x, ԭ��_y);
	cv::Point ��_2(locat[0] * ���� + ԭ��_x, -locat[1] * ����  + ԭ��_y);
	cv::line(ͼ��, ��_1, ��_2, ��ɫ, ��ϸ);
}

__host__ __device__ vector operator+(vector ����_1, vector ����_2)
{
	return vector(����_1[0] + ����_2[0], ����_1[1] + ����_2[1]);
}

__host__ __device__ vector operator-(vector ����_1, vector ����_2)
{
	return vector(����_1[0] - ����_2[0], ����_1[1] - ����_2[1]);
}

__host__ __device__ vector operator*(vector ����, double ��)
{
	return vector(����[0] * ��, ����[1] * ��);
}

__host__ __device__ vector operator*(double ��, vector ����)
{
	return vector(����[0] * ��, ����[1] * ��);
}

__host__ __device__ vector operator/(vector ����, double ��)
{
	return vector(����[0] / ��, ����[1] / ��);
}


__host__ __device__ double length(vector ����)
{
	return length({ 0,0 }, ����);
}



__host__ __device__ double operator*(vector ����_1, vector ����_2)
{
	return ����_1[0] * ����_2[0] + ����_1[1] * ����_2[1];
}

__host__ __device__ double operator^(vector ����_1, vector ����_2)
{
	return (����_1[0] * ����_2[1]) - (����_1[1] * ����_2[0]);
}