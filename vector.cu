#include "hip/hip_runtime.h"
#include "geometry.cuh"


__host__ __device__ vector::vector() :point(1, 0) {}

__host__ __device__ vector::vector(double x, double y) : point(x, y) {}

__host__ __device__ vector::vector(point ��) : point(��) {};

__host__ __device__ vector::vector(double ����[2], double ����)
{
	double ���� = ���� / length({ 0,0 }, ����);
	locat[0] = ����[0] * ����;
	locat[1] = ����[1] * ����;
}

__host__ __device__ vector::vector(double �Ƕ�, bool rad, double ����)
{
	if (!rad)
	{
		�Ƕ� = deg2rad(�Ƕ�);
	}
	locat[0] = cos(�Ƕ�) * ����;
	locat[1] = sin(�Ƕ�) * ����;
}

__host__ __device__ double& vector::operator[](int i)
{
	return locat[i & 1];
}

__host__ __device__ vector& vector::operator+=(vector ����)
{
	locat[0] += ����[0];
	locat[1] += ����[1];
	return *this;
}

__host__ __device__ vector& vector::operator-=(vector ����)
{
	locat[0] -= ����[0];
	locat[1] -= ����[1];
	return *this;
}

__host__ __device__ vector& vector::operator*=(double ��)
{
	locat[0] *= ��;
	locat[1] *= ��;
	return *this;
}

__host__ __device__ vector& vector::operator/=(double ��)
{
	locat[0] /= ��;
	locat[1] /= ��;
	return *this;
}

__host__ __device__ vector vector::unitize()
{
	return vector(*this / length(*this));
}

__host__ __device__ vector operator+(vector ����_1, vector ����_2)
{
	return vector(����_1[0] + ����_2[0], ����_1[1] + ����_2[1]);
}

__host__ __device__ vector operator-(vector ����_1, vector ����_2)
{
	return vector(����_1[0] - ����_2[0], ����_1[1] - ����_2[1]);
}

__host__ __device__ vector operator*(vector ����, double ��)
{
	return vector(����[0] * ��, ����[1] * ��);
}

__host__ __device__ vector operator*(double ��, vector ����)
{
	return vector(����[0] * ��, ����[1] * ��);
}

__host__ __device__ vector operator/(vector ����, double ��)
{
	return vector(����[0] / ��, ����[1] / ��);
}