
#include <hip/hip_runtime.h>
//#include "geometry.cuh"
//
//#define _USE_MATH_DEFINES 
//#include <math.h>
//
//__host__ __device__ triangle::triangle()
//{
//	segs[0].origin = point(0, 0);
//	segs[0].dir = vector(point(1, 0));
//	segs[0].dist = 1;
//	segs[1].origin = point(1, 0);
//	segs[1].dir = vector(point(-M_SQRT1_2, M_SQRT1_2));
//	segs[1].dist = M_SQRT2;
//	segs[2].origin = point(0, 1);
//	segs[2].dir = vector(point(0, -1));
//	segs[2].dist = 1;
//}
//
//__host__ __device__ triangle::triangle(point* ��)
//{
//	segs[0] = seg(��[0], ��[1]);
//	segs[1] = seg(��[1], ��[2]);
//	segs[2] = seg(��[2], ��[0]);
//}
//
//__host__ __device__ triangle::triangle(seg* �߶�)
//{
//	segs[0] = �߶�[0];
//	segs[1] = �߶�[1];
//	segs[2] = �߶�[2];
//}
