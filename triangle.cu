
#include <hip/hip_runtime.h>
//#include "geometry.cuh"
//
//#define _USE_MATH_DEFINES 
//#include <math.h>
//
//__host__ __device__ triangle::triangle()
//{
//	segs[0].origin = point(0, 0);
//	segs[0].dir = vector(point(1, 0));
//	segs[0].dist = 1;
//	segs[1].origin = point(1, 0);
//	segs[1].dir = vector(point(-M_SQRT1_2, M_SQRT1_2));
//	segs[1].dist = M_SQRT2;
//	segs[2].origin = point(0, 1);
//	segs[2].dir = vector(point(0, -1));
//	segs[2].dist = 1;
//}
//
//__host__ __device__ triangle::triangle(point* 点)
//{
//	segs[0] = seg(点[0], 点[1]);
//	segs[1] = seg(点[1], 点[2]);
//	segs[2] = seg(点[2], 点[0]);
//}
//
//__host__ __device__ triangle::triangle(seg* 线段)
//{
//	segs[0] = 线段[0];
//	segs[1] = 线段[1];
//	segs[2] = 线段[2];
//}
