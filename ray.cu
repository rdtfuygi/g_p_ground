#include "hip/hip_runtime.h"
#include "geometry.cuh"

__host__ __device__ ray::ray() :line() {}

__host__ __device__ ray::ray(point ��, vector ����) :line(��, ����) {}

__host__ __device__ ray::ray(point ��, double �Ƕ�, bool rad) :line(��, �Ƕ�, rad) {}

__host__ __device__ ray::ray(double ��_1_x, double ��_1_y, double ��_2_x, double ��_2_y) :line(��_1_x, ��_1_y, ��_2_x, ��_2_y) {}

__host__ __device__ ray::ray(point ��_1, point ��_2) :line(��_1, ��_2) {}

__host__ __device__ ray ray::rotate(const point ��, double �Ƕ�, bool rad) const
{
	return ray(::rotate(��, origin, �Ƕ�, rad), dir.rotate(�Ƕ�, rad));
}

__host__ __device__ double ray::point_dist(const point ��) const
{
	line temp;
	temp.origin = ��;
	temp.dir[0] = dir[1];
	temp.dir[1] = -dir[0];

	double t_1, t_2;
	cross(*this, temp, t_1, t_2);
	if (t_1 > 0)
	{
		return abs(t_2);
	}
	else
	{
		return length(��, origin);
	}
}


__host__ __device__ void cross(const ray l_1, const line l_2, double& t_1, double& t_2)
{
	cross(line(l_1), line(l_2), t_1, t_2);
	if (0 > t_1)
	{
		t_1 = DBL_MAX;
		t_2 = DBL_MAX;
	}
}

__host__ __device__ void cross(const ray l_1, const ray l_2, double& t_1, double& t_2)
{
	if (((l_1.origin[0] < l_2.origin[0]) && (l_1.dir[0] < 0) && (0 < l_2.dir[0])) ||
		((l_1.origin[0] > l_2.origin[0]) && (l_1.dir[0] > 0) && (0 > l_2.dir[0])) ||
		((l_1.origin[1] < l_2.origin[1]) && (l_1.dir[1] < 0) && (0 < l_2.dir[1])) ||
		((l_1.origin[1] > l_2.origin[1]) && (l_1.dir[1] > 0) && (0 > l_2.dir[1])))
	{
		t_1 = DBL_MAX;
		t_2 = DBL_MAX;
		return;
	}
	cross(line(l_1), line(l_2), t_1, t_2);
	if ((0 > t_1) || (0 > t_2))
	{
		t_1 = DBL_MAX;
		t_2 = DBL_MAX;
	}
}

__host__ __device__ void cross(const ray l_1, const seg l_2, double& t_1, double& t_2)
{
	point end = l_2.end();
	if ((fmin(l_2.origin[0], end[0]) > l_1.origin[0]) && (l_2.dir[0] < 0) ||
		(fmin(l_2.origin[1], end[1]) > l_1.origin[1]) && (l_2.dir[1] < 0) ||
		(fmax(l_2.origin[0], end[0]) < l_1.origin[0]) && (l_2.dir[0] > 0) ||
		(fmax(l_2.origin[1], end[1]) < l_1.origin[1]) && (l_2.dir[1] > 0))
	{
		t_1 = DBL_MAX;
		t_2 = DBL_MAX;
		return;
	}
	cross(line(l_1), line(l_2), t_1, t_2);
	if ((0 > t_1) || (0 > t_2) || (t_2 > l_2.dist))
	{
		t_1 = DBL_MAX;
		t_2 = DBL_MAX;
	}
}

__host__ __device__ point cross(const ray l_1, const line l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return l_1.point_get(t_1);
	}
	else
	{
		return point(DBL_MAX, DBL_MAX);
	}
}

__host__ __device__ point cross(const ray l_1, const ray l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return l_1.point_get(t_1);
	}
	else
	{
		return point(DBL_MAX, DBL_MAX);
	}
}

__host__ __device__ point cross(const ray l_1, const seg l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return l_1.point_get(t_1);
	}
	else
	{
		return point(DBL_MAX, DBL_MAX);
	}
}

__host__ __device__ bool is_cross(const ray l_1, const seg l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return true;
	}
	return false;
}


