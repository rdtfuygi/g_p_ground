#include "hip/hip_runtime.h"
#include "geometry.cuh"

__host__ __device__ seg::seg() :ray(), dist(1) {}

__host__ __device__ seg::seg(point ��, vector ����, double ����) : ray(��, ����), dist(����) {}

__host__ __device__ seg::seg(point ��, double ����, double ����, bool rad) : ray(��, ����, rad), dist(����) {}

__host__ __device__ seg::seg(double ��_1_x, double ��_1_y, double ��_2_x, double ��_2_y) :ray(��_1_x, ��_1_y, ��_2_x, ��_2_y), dist(length(��_1_x, ��_1_y, ��_2_x, ��_2_y)) {}

__host__ __device__ seg::seg(point ��_1, point ��_2) : ray(��_1, ��_2), dist(length(��_1, ��_2)) {}

__host__ __device__ point seg::end() const
{
	return point(vector(origin) + (dir * dist));
}

__host__ __device__ seg seg::rotate(const point ��, double �Ƕ�, bool rad) const
{
	return seg(::rotate(��, origin, �Ƕ�, rad), dir.rotate(�Ƕ�, rad));
}

__host__ __device__ double seg::point_dist(const point ��) const
{
	line temp;
	temp.origin = ��;
	temp.dir[0] = dir[1];
	temp.dir[1] = -dir[0];

	double t_1, t_2;
	cross(*this, temp, t_1, t_2);
	if (t_1 < 0)
	{
		return length(��, origin);
	}
	else if (t_1 > dist)
	{
		return length(��, end());
	}
	else
	{
		return abs(t_2);
	}
}

void seg::print(cv::InputOutputArray ͼ��, double ����, const cv::Scalar& ��ɫ, int ��ϸ) const
{
	int �� = ͼ��.rows(), �� = ͼ��.cols();
	int ԭ��_x = �� / 2, ԭ��_y = �� / 2;

	int �Ŵ� = 2 * (�� > �� ? �� : ��);

	cv::Point ��_1(origin[0] * ���� + ԭ��_x, -origin[1] * ���� + ԭ��_y);
	cv::Point ��_2(end()[0] * ���� + ԭ��_x, -end()[1] * ���� + ԭ��_y);
	cv::line(ͼ��, ��_1, ��_2, ��ɫ, ��ϸ);
}


__host__ __device__ void cross(const seg l_1, const line l_2, double& t_1, double& t_2)
{
	cross(line(l_1), line(l_2), t_1, t_2);
	if ((0 > t_1) || (t_1 > l_1.dist))
	{
		t_1 = DBL_MAX;
		t_2 = DBL_MAX;
	}
}

__host__ __device__ void cross(const seg l_1, const ray l_2, double& t_1, double& t_2)
{
	point end = l_1.end();
	if ((fmin(l_1.origin[0], end[0]) > l_2.origin[0]) && (l_1.dir[0] < 0) ||
		(fmin(l_1.origin[1], end[1]) > l_2.origin[1]) && (l_1.dir[1] < 0) ||
		(fmax(l_1.origin[0], end[0]) < l_2.origin[0]) && (l_1.dir[0] > 0) ||
		(fmax(l_1.origin[1], end[1]) < l_2.origin[1]) && (l_1.dir[1] > 0))
	{
		t_2 = DBL_MAX;
		t_1 = DBL_MAX;
		return;
	}
	cross(line(l_2), line(l_1), t_2, t_1);
	if ((0 > t_2) || (0 > t_1) || (t_1 > l_1.dist))
	{
		t_2 = DBL_MAX;
		t_1 = DBL_MAX;
	}
}

__host__ __device__ void cross(const seg l_1, const seg l_2, double& t_1, double& t_2)
{
	point end_1 = l_1.end();
	point end_2 = l_2.end();
	if ((fmin(l_1.origin[0], end_1[0]) > fmax(l_2.origin[0], end_2[0])) ||
		(fmin(l_2.origin[1], end_2[1]) > fmax(l_1.origin[1], end_1[1])) ||
		(fmin(l_1.origin[0], end_1[0]) > fmax(l_2.origin[0], end_2[0])) ||
		(fmin(l_2.origin[1], end_2[1]) > fmax(l_1.origin[1], end_1[1])))
	{
		t_2 = DBL_MAX;
		t_1 = DBL_MAX;
		return;
	}
	cross(line(l_1), line(l_2), t_2, t_1);
	if ((0 > t_2) || (t_2 > l_2.dist) || (0 > t_1) || (t_1 > l_1.dist))
	{
		t_2 = DBL_MAX;
		t_1 = DBL_MAX;
	}
}

__host__ __device__ point cross(const seg l_1, const line l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return l_1.point_get(t_1);
	}
	else
	{
		return point(DBL_MAX, DBL_MAX);
	}
}

__host__ __device__ point cross(const seg l_1, const ray l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return l_1.point_get(t_1);
	}
	else
	{
		return point(DBL_MAX, DBL_MAX);
	}
}

__host__ __device__ point cross(const seg l_1, const seg l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return l_1.point_get(t_1);
	}
	else
	{
		return point(DBL_MAX, DBL_MAX);
	}
}

__host__ __device__ bool is_cross(const seg l_1, const seg l_2)
{
	vector a = vector(l_2.origin) - vector(l_1.origin);
	vector b = vector(l_2.end()) - vector(l_1.origin);
	vector c = vector(l_1.origin) - vector(l_2.origin);
	vector d = vector(l_1.end()) - vector(l_2.origin);
	return ((a ^ b) < 0) && ((c ^ d) < 0);
}

__host__ __device__ bool is_cross(const seg l_1, const ray l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return true;
	}
	return false;
}