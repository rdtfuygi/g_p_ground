#include "hip/hip_runtime.h"
#include "geometry.cuh"
#include <numbers>
#include <limits>
#define _USE_MATH_DEFINES 
#include <math.h>

__host__ __device__ double deg2rad(double rad)
{
	return rad / M_PI * 180;
}

__host__ __device__ double rad2deg(double deg)
{
	return deg * M_PI / 180;
}



__host__ __device__ double length(double ��_1_x, double ��_1_y, double ��_2_x, double ��_2_y)
{
	return sqrt(pow(��_1_x - ��_2_x, 2) + pow(��_1_y - ��_2_y, 2));
}

__host__ __device__ double length(point ��_1, point ��_2)
{
	return length(��_1[0], ��_1[1], ��_2[0], ��_2[1]);
}

__host__ __device__ point rotate(const point ��_1, const point ��_2, double �Ƕ�, bool rad)
{
	if (!rad)
	{
		�Ƕ� = deg2rad(�Ƕ�);
	}
	double x = (��_2[0] - ��_1[0]) * cos(�Ƕ�) - (��_2[1] - ��_1[1]) * sin(�Ƕ�) + ��_1[0];
	double y = (��_2[0] - ��_1[0]) * sin(�Ƕ�) + (��_2[1] - ��_1[1]) * cos(�Ƕ�) + ��_1[1];
	return point(x, y);
}
