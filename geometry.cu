#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES 

#include "geometry.cuh"
#include <limits>


__host__ __device__ double deg2rad(double rad)
{
	return rad * M_PI / 180;
}

__host__ __device__ double rad2deg(double deg)
{
	return deg / M_PI * 180;
}



__host__ __device__ double length(double ��_1_x, double ��_1_y, double ��_2_x, double ��_2_y)
{
	return sqrt(pow(��_1_x - ��_2_x, 2) + pow(��_1_y - ��_2_y, 2));
}

__host__ __device__ double length(point ��_1, point ��_2)
{
	return length(��_1[0], ��_1[1], ��_2[0], ��_2[1]);
}

__host__ __device__ point rotate(const point ԭ��, const point ��_2, double �Ƕ�, bool rad)
{
	if (!rad)
	{
		�Ƕ� = deg2rad(�Ƕ�);
	}
	double x = (��_2[0] - ԭ��[0]) * cos(�Ƕ�) - (��_2[1] - ԭ��[1]) * sin(�Ƕ�) + ԭ��[0];
	double y = (��_2[0] - ԭ��[0]) * sin(�Ƕ�) + (��_2[1] - ԭ��[1]) * cos(�Ƕ�) + ԭ��[1];
	return point(x, y);
}
