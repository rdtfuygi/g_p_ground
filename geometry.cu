#include "hip/hip_runtime.h"
#include "geometry.cuh"
#include <numbers>










__host__ __device__ double deg2rad(double rad)
{
	return rad / std::numbers::pi * 180;
}

__host__ __device__ double rad2deg(double deg)
{
	return deg * std::numbers::pi / 180;
}



__host__ __device__ double length(double ��_1_x, double ��_1_y, double ��_2_x, double ��_2_y)
{
	return sqrt(pow(��_1_x - ��_2_x, 2) + pow(��_1_y - ��_2_y, 2));
}

__host__ __device__ double length(point ��_1, point ��_2)
{
	return length(��_1[0], ��_1[1], ��_2[0], ��_2[1]);
}

__host__ __device__ double length(vector ����)
{
	return length({ 0,0 }, ����);
}



__host__ __device__ double operator*(vector ����_1, vector ����_2)
{
	return ����_1[0] * ����_2[0] + ����_1[1] * ����_2[1];
}

__host__ __device__ double operator^(vector ����_1, vector ����_2)
{
	return __host__ __device__ double();
}