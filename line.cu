#include "hip/hip_runtime.h"
#include "geometry.cuh"

__host__ __device__ line::line() :origin(), dir() {}

__host__ __device__ line::line(point ��, vector ����) : origin(��), dir(����.unitize()) {}

__host__ __device__ line::line(point ��, double �Ƕ�, bool rad) : origin(��)
{
	if (!rad)
	{
		�Ƕ� = deg2rad(�Ƕ�);
	}
	dir[0] = cos(�Ƕ�);
	dir[1] = sin(�Ƕ�);
}

__host__ __device__ line::line(double ��_1_x, double ��_1_y, double ��_2_x, double ��_2_y) :origin(��_1_x, ��_1_y), dir((vector(��_2_x, ��_2_y) - vector(��_1_x, ��_1_y)).unitize()) {}

__host__ __device__ line::line(point ��_1, point ��_2) :origin(��_1), dir((vector(��_2) - vector(��_1)).unitize()) {}

__host__ __device__ line::line(double k, double b) :origin(0, b), dir(vector(1, k + b).unitize()) {}

__host__ __device__ point line::point_get(double t) const
{
	return point(vector(origin) + (dir * t));
}

__host__ __device__ double line::angle_get(bool rad) const
{
	return dir.angle_get(rad);
}

__host__ __device__ line line::rotate(const point ��, double �Ƕ�, bool rad) const
{
	return line(::rotate(��, origin, �Ƕ�, rad), dir.rotate(�Ƕ�, rad));
}

__host__ __device__ double line::point_dist(const point ��) const
{
	line temp;
	temp.origin = ��;
	temp.dir[0] = dir[1];
	temp.dir[1] = -dir[0];

	double t_1, t_2;
	cross(*this, temp, t_1, t_2);
	return abs(t_2);
}


__host__ __device__ void cross(const line l_1, const line l_2, double& t_1, double& t_2)
{
	double ����[2][3] =
	{
		{l_1.dir[0],-l_2.dir[0],-l_1.origin[0] + l_2.origin[0]},
		{l_1.dir[1],-l_2.dir[1],-l_1.origin[1] + l_2.origin[1]}
	};
	if (����[0][0] != 0)
	{
		{
			double a00 = ����[0][0];
			double a10 = ����[1][0];
			for (int i = 0; i < 3; i++)
			{
				����[0][i] /= a00;
				����[1][i] -= ����[0][i] * a10;
			}
		}

		if (����[1][1] == 0)
		{
			t_1 = DBL_MAX;
			t_2 = DBL_MAX;
			return;
		}

		{
			double a01 = ����[0][1];
			double a11 = ����[1][1];
			for (int i = 0; i < 3; i++)
			{
				����[1][i] /= a11;
				����[0][i] -= ����[1][i] * a01;
			}
		}

		t_1 = ����[0][2];
		t_2 = ����[1][2];
	}
	else if (����[1][0] != 0)
	{
		{
			double a10 = ����[1][0];
			double a00 = ����[0][0];
			for (int i = 0; i < 3; i++)
			{
				����[1][i] /= a10;
				����[0][i] -= ����[1][i] * a00;
			}
		}

		if (����[0][1] == 0)
		{
			t_1 = DBL_MAX;
			t_2 = DBL_MAX;
			return;
		}

		{
			double a11 = ����[1][1];
			double a01 = ����[0][1];
			for (int i = 0; i < 3; i++)
			{
				����[0][i] /= a01;
				����[1][i] -= ����[0][i] * a11;
			}
		}


		t_2 = ����[0][2];
		t_1 = ����[1][2];
	}
	else
	{
		t_1 = DBL_MAX;
		t_2 = DBL_MAX;
	}
}

__host__ __device__ void cross(const line l_1, const ray l_2, double& t_1, double& t_2)
{
	cross(line(l_1), line(l_2), t_1, t_2);
	if (0 > t_2)
	{
		t_1 = DBL_MAX;
		t_2 = DBL_MAX;
	}
}

__host__ __device__ void cross(const line l_1, const seg l_2, double& t_1, double& t_2)
{
	cross(line(l_1), line(l_2), t_1, t_2);
	if ((0 > t_2) || (t_2 > l_2.dist))
	{
		t_1 = DBL_MAX;
		t_2 = DBL_MAX;
	}
}

__host__ __device__ point cross(const line l_1, const line l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return l_1.point_get(t_1);
	}
	else
	{
		return point(DBL_MAX, DBL_MAX);
	}
}

__host__ __device__ point cross(const line l_1, const ray l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return l_1.point_get(t_1);
	}
	else
	{
		return point(DBL_MAX, DBL_MAX);
	}
}

__host__ __device__ point cross(const line l_1, const seg l_2)
{
	double t_1, t_2;
	cross(l_1, l_2, t_1, t_2);
	if (t_1 != DBL_MAX)
	{
		return l_1.point_get(t_1);
	}
	else
	{
		return point(DBL_MAX, DBL_MAX);
	}
}