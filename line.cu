#include "hip/hip_runtime.h"
#include "geometry.cuh"

__host__ __device__ line::line() :origin(), dir() {}

__host__ __device__ line::line(point ��, vector ����) : origin(��), dir(����.unitize()) {}

__host__ __device__ line::line(point ��, double �Ƕ�, bool rad) : origin(��)
{
	if (!rad)
	{
		�Ƕ� = deg2rad(�Ƕ�);
	}
	dir[0] = cos(�Ƕ�);
	dir[1] = sin(�Ƕ�);
}

__host__ __device__ line::line(double ��_1_x, double ��_1_y, double ��_2_x, double ��_2_y) :origin(��_1_x, ��_1_y), dir((vector(��_2_x, ��_2_y) - vector(��_1_x, ��_1_y)).unitize()) {}

__host__ __device__ line::line(point ��_1, point ��_2) :origin(��_1), dir((vector(��_2) - vector(��_1)).unitize()) {}

__host__ __device__ line::line(double k, double b) :origin(0, b), dir(vector(1, k + b).unitize()) {}

__host__ __device__ point line::point_get(double t)
{
	return point(vector(origin) + (dir * t));
}
